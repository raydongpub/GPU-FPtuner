#include "hip/hip_runtime.h"
//#include <hip/hip_runtime_api.h>

#include <helper_timer.h>

#include <mpfr.h>

#include <qd/dd_real.h>

#include "../../gpuprec/gqd/gqd.cu"

using namespace std;

typedef struct gdd_real3

{

	gdd_real x, y, z;

} gdd_real3;

typedef struct dd_real3 

{

  dd_real x, y, z;

} dd_real3;

void qd2gqd(dd_real3* dd_data, gdd_real3* gdd_data, const unsigned int numElement) {

    for (unsigned int i = 0; i < numElement; i++) {

        gdd_data[i].x.x = dd_data[i].x.x[0];

        gdd_data[i].x.y = dd_data[i].x.x[1];

        gdd_data[i].y.x = dd_data[i].y.x[0];

        gdd_data[i].y.y = dd_data[i].y.x[1];

        gdd_data[i].z.x = dd_data[i].z.x[0];

        gdd_data[i].z.y = dd_data[i].z.x[1];

    }

}

void qd2gqd(dd_real* dd_data, gdd_real* gdd_data, const unsigned int numElement) {

    for (unsigned int i = 0; i < numElement; i++) {

        gdd_data[i].x = dd_data[i].x[0];

        gdd_data[i].y = dd_data[i].x[1];

    }

}

void gqd2qd(gdd_real* gdd_data, dd_real* dd_data, const unsigned int numElement) {

    for (unsigned int i = 0; i < numElement; i++) {

        dd_data[i].x[0] = gdd_data[i].x;

        dd_data[i].x[1] = gdd_data[i].y;

    }

}

// Copyright 2009, Andrew Corrigan, acorriga@gmu.edu

// This code is from the AIAA-2009-4001 paper

// #include <cutil.h>

//#include <hip/hip_runtime_api.h>

//#include <helper_timer.h>

#include <iostream>

#include <fstream>

#include <math.h>

#include <hip/hip_runtime_api.h>

#include <sys/time.h>

//#include <qd/dd_real.h>

//#include "gpuprec/gqd/gqd.cu"

//#if CUDART_VERSION < 3000

//struct gdd_real3

//{

//	gdd_real x, y, z;

//};

//#endif

/*

 * Options 

 * 

 */

//typedef struct gdd_real3

//{

//	gdd_real x, y, z;

//};

//typedef struct dd_real3

//{

//	dd_real x, y, z;

//};

#define GAMMA 1.4

#define iterations 2000

#ifndef block_length

	#define block_length 128

#endif

#define NDIM 3

#define NNB 4

/* (previously processed: ignoring self-referential macro declaration) macro name = RK */ 

#define ff_mach 1.2

#define deg_angle_of_attack 0.0

template < typename T >

void check ( T result, char const * const func, const char * const file,

           int const line ) {

  if ( result ) {

    fprintf ( stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,

            static_cast < unsigned int > ( result ), _cudaGetErrorEnum ( result ), func );

    exit ( 1 );

  }

}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

static const char *_cudaGetErrorEnum(hipError_t error)

{

  return hipGetErrorName(error);

}

/*

 * not options

 */

#if block_length > 128

#warning "the kernels may fail too launch on some systems if the block length is too large"

#endif

#define VAR_DENSITY 0

#define VAR_MOMENTUM  1

#define VAR_DENSITY_ENERGY (VAR_MOMENTUM+NDIM)

#define NVAR (VAR_DENSITY_ENERGY+1)

/*

 * Generic functions

 */

template < typename T >

T * alloc ( int N )

{

 T * t;

 check ( ( hipMalloc ( ( void * * ) & t, sizeof ( T ) * N ) ), "hipMalloc((void**)&t, sizeof(T)*N)", "euler3d_gdd_real.cu", 86 );

 return t;

}

template < typename T >

void dealloc ( T * array )

{

 check ( ( hipFree ( ( void * ) array ) ), "hipFree((void*)array)", "euler3d_gdd_real.cu", 93 );

}

template < typename T >

void copy ( T * dst, T * src, int N )

{

 check ( ( hipMemcpy ( ( void * ) dst, ( void * ) src, N * sizeof ( T ), hipMemcpyDeviceToDevice ) ), "hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToDevice)", "euler3d_gdd_real.cu", 99 );

}

template < typename T >

void upload ( T * dst, T * src, int N )

{

 check ( ( hipMemcpy ( ( void * ) dst, ( void * ) src, N * sizeof ( T ), hipMemcpyHostToDevice ) ), "hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyHostToDevice)", "euler3d_gdd_real.cu", 105 );

}

template < typename T >

void download ( T * dst, T * src, int N )

{

 check ( ( hipMemcpy ( ( void * ) dst, ( void * ) src, N * sizeof ( T ), hipMemcpyDeviceToHost ) ), "hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToHost)", "euler3d_gdd_real.cu", 111 );

}

//void dump(gdd_real* variables, int nel, int nelr)

//{

//	gdd_real* h_variables = new double[nelr*NVAR];

//	download(h_variables, variables, nelr*NVAR);

//  gdd_real output;

//	{

//		std::ofstream file("density");

//		file << nel << " " << nelr << std::endl;

//		for(int i = 0; i < nel; i++) {

//      output = h_variables[i + VAR_DENSITY*nelr];

//      file << output << std::endl;

//    }

//	}

//

//

//	{

//		std::ofstream file("momentum");

//		file << nel << " " << nelr << std::endl;

//		for(int i = 0; i < nel; i++)

//		{

//			for(int j = 0; j != NDIM; j++) {

//        output = h_variables[i + (VAR_MOMENTUM+j)*nelr];

//				file << output << " ";

//      }

//			file << std::endl;

//		}

//	}

//	

//	{

//		std::ofstream file("density_energy");

//		file << nel << " " << nelr << std::endl;

//		for(int i = 0; i < nel; i++) {

//      output = h_variables[i + VAR_DENSITY_ENERGY*nelr];

//      file << output << std::endl;

//    }

//	}

//	delete[] h_variables;

//}

/*

 * Element-based Cell-centered FVM solver functions

 */

__constant__ float ff_variable[5];

__constant__ float3 ff_flux_contribution_momentum_x[1];

__constant__ float3 ff_flux_contribution_momentum_y[1];

__constant__ float3 ff_flux_contribution_momentum_z[1];

__constant__ float3 ff_flux_contribution_density_energy[1];

__global__ void cuda_initialize_variables(int nelr,float *variables)

{

  const int i = (blockDim . x * blockIdx . x + threadIdx . x);

  for (int j = 0; j < 1 + 3 + 1; j++) 

    variables[i + j * nelr] = ff_variable[j];

}

__global__ void cuda_initialize_variables_1(int nelr,float *variables)

{

  const int i = (blockDim . x * blockIdx . x + threadIdx . x);

  for (int j = 0; j < 1 + 3 + 1; j++) 

    variables[i + j * nelr] = ff_variable[j];

}

__global__ void cuda_initialize_variables_2(int nelr,float *variables)

{

  const int i = (blockDim . x * blockIdx . x + threadIdx . x);

  for (int j = 0; j < 1 + 3 + 1; j++) 

    variables[i + j * nelr] = ff_variable[j];

}

void initialize_variables(int nelr,float *variables)

{

  ::dim3 Dg((nelr / 128));

  ::dim3 Db(128);

  cuda_initialize_variables<<<Dg,Db>>>(nelr,variables);

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {

    fprintf(stderr,"GPUassert: %s Initializing variables\n",(hipGetErrorString(error)));

    exit(- 1);

  }

}

void initialize_variables_1(int nelr,float *variables)

{

  ::dim3 Dg((nelr / 128));

  ::dim3 Db(128);

  cuda_initialize_variables_1<<<Dg,Db>>>(nelr,variables);

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {

    fprintf(stderr,"GPUassert: %s Initializing variables\n",(hipGetErrorString(error)));

    exit(- 1);

  }

}

void initialize_variables_2(int nelr,float *variables)

{

  ::dim3 Dg((nelr / 128));

  ::dim3 Db(128);

  cuda_initialize_variables_2<<<Dg,Db>>>(nelr,variables);

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {

    fprintf(stderr,"GPUassert: %s Initializing variables\n",(hipGetErrorString(error)));

    exit(- 1);

  }

}

inline __device__ void compute_flux_contribution(float &density,float3 &momentum,float &dens_energy,float &pressure,float3 &velocity,float3 &fc_momentum_x,float3 &fc_momentum_y,float3 &fc_momentum_z,float3 &fc_density_energy)

{

  fc_momentum_x . x = velocity . x * momentum . x + pressure;

  fc_momentum_x . y = velocity . x * momentum . y;

  fc_momentum_x . z = velocity . x * momentum . z;

  fc_momentum_y . x = fc_momentum_x . y;

  fc_momentum_y . y = velocity . y * momentum . y + pressure;

  fc_momentum_y . z = velocity . y * momentum . z;

  fc_momentum_z . x = fc_momentum_x . z;

  fc_momentum_z . y = fc_momentum_y . z;

  fc_momentum_z . z = velocity . z * momentum . z + pressure;

  float de_p = dens_energy + pressure;

  fc_density_energy . x = velocity . x * de_p;

  fc_density_energy . y = velocity . y * de_p;

  fc_density_energy . z = velocity . z * de_p;

}

inline __device__ void compute_flux_contribution_1(float &density,float3 &momentum,float &dens_energy,float &pressure,float3 &velocity,float3 &fc_momentum_x,float3 &fc_momentum_y,float3 &fc_momentum_z,float3 &fc_density_energy)

{

  fc_momentum_x . x = velocity . x * momentum . x + pressure;

  fc_momentum_x . y = velocity . x * momentum . y;

  fc_momentum_x . z = velocity . x * momentum . z;

  fc_momentum_y . x = fc_momentum_x . y;

  fc_momentum_y . y = velocity . y * momentum . y + pressure;

  fc_momentum_y . z = velocity . y * momentum . z;

  fc_momentum_z . x = fc_momentum_x . z;

  fc_momentum_z . y = fc_momentum_y . z;

  fc_momentum_z . z = velocity . z * momentum . z + pressure;

  float de_p = dens_energy + pressure;

  fc_density_energy . x = velocity . x * de_p;

  fc_density_energy . y = velocity . y * de_p;

  fc_density_energy . z = velocity . z * de_p;

}

void compute_flux_contribution_host(float &density,float3 &momentum,float &dens_energy,float &pressure,float3 &velocity,float3 &fc_momentum_x,float3 &fc_momentum_y,float3 &fc_momentum_z,float3 &fc_density_energy)

{

  fc_momentum_x . x = velocity . x * momentum . x + pressure;

  fc_momentum_x . y = velocity . x * momentum . y;

  fc_momentum_x . z = velocity . x * momentum . z;

  fc_momentum_y . x = fc_momentum_x . y;

  fc_momentum_y . y = velocity . y * momentum . y + pressure;

  fc_momentum_y . z = velocity . y * momentum . z;

  fc_momentum_z . x = fc_momentum_x . z;

  fc_momentum_z . y = fc_momentum_y . z;

  fc_momentum_z . z = velocity . z * momentum . z + pressure;

  float de_p = dens_energy + pressure;

  fc_density_energy . x = velocity . x * de_p;

  fc_density_energy . y = velocity . y * de_p;

  fc_density_energy . z = velocity . z * de_p;

}

inline __device__ void compute_velocity(float &density,float3 &momentum,float3 &velocity)

{

  velocity . x =( momentum . x / density);

  velocity . y =( momentum . y / density);

  velocity . z =( momentum . z / density);

}

inline __device__ void compute_velocity_1(float &density,float3 &momentum,float3 &velocity)

{

  velocity . x =( momentum . x / density);

  velocity . y =( momentum . y / density);

  velocity . z =( momentum . z / density);

}

inline __device__ void compute_velocity_2(float &density,float3 &momentum,float3 &velocity)

{

  velocity . x =( momentum . x / density);

  velocity . y =( momentum . y / density);

  velocity . z =( momentum . z / density);

}

inline __device__ float compute_speed_sqd(float3 &velocity)

{

  return (float)(velocity . x * velocity . x + velocity . y * velocity . y + velocity . z * velocity . z);

}

inline __device__ float compute_speed_sqd_1(float3 &velocity)

{

  return (float)(velocity . x * velocity . x + velocity . y * velocity . y + velocity . z * velocity . z);

}

inline __device__ float compute_speed_sqd_2(float3 &velocity)

{

  return (float)(velocity . x * velocity . x + velocity . y * velocity . y + velocity . z * velocity . z);

}

inline __device__ float compute_pressure(float &density,float &density_energy,float &speed_sqd)

{

  return (float)((((double )1.4) - ((double )1.0)) * (density_energy - ((double )0.5) * density * speed_sqd));

}

inline __device__ float compute_pressure_1(float &density,float &density_energy,float &speed_sqd)

{

  return (float)((((double )1.4) - ((double )1.0)) * (density_energy - ((double )0.5) * density * speed_sqd));

}

inline __device__ float compute_pressure_2(float &density,float &density_energy,float &speed_sqd)

{

  return (float)((((double )1.4) - ((double )1.0)) * (density_energy - ((double )0.5) * density * speed_sqd));

}

inline __device__ float compute_speed_of_sound(float &density,float &pressure)

{

  return (float)(sqrt(((double )1.4) * pressure / density));

}

inline __device__ float compute_speed_of_sound_1(float &density,float &pressure)

{

  return (float)(sqrt(((double )1.4) * pressure / density));

}

inline __device__ float compute_speed_of_sound_2(float &density,float &pressure)

{

  return (float)(sqrt(((double )1.4) * pressure / density));

}

__global__ void cuda_compute_step_factor(int nelr,float *variables,float *areas,float *step_factors)

{

  const int i = (blockDim . x * blockIdx . x + threadIdx . x);

  float density =( variables[i + 0 * nelr]);

  float3 momentum;

  momentum . x =( variables[i + (1 + 0) * nelr]);

  momentum . y =( variables[i + (1 + 1) * nelr]);

  momentum . z =( variables[i + (1 + 2) * nelr]);

  float density_energy =( variables[i + (1 + 3) * nelr]);

  float3 velocity;

  compute_velocity(density,momentum,velocity);

  float speed_sqd = compute_speed_sqd(velocity);

  float pressure = compute_pressure(density,density_energy,speed_sqd);

  float speed_of_sound = compute_speed_of_sound(density,pressure);

// dt = double(0.5) * sqrt(areas[i]) /  (||v|| + c).... but when we do time stepping, this later would need to be divided by the area, so we just do it all at once

  step_factors[i] = ((double )0.5) / (sqrt(areas[i]) * (sqrt(speed_sqd) + speed_of_sound));

}

void compute_step_factor(int nelr,float *variables,float *areas,float *step_factors)

{

  ::dim3 Dg((nelr / 128));

  ::dim3 Db(128);

  cuda_compute_step_factor<<<Dg,Db>>>(nelr,variables,areas,step_factors);

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {

    fprintf(stderr,"GPUassert: %s compute_step_factor failed\n",(hipGetErrorString(error)));

    exit(- 1);

  }

}

/*

 *

 *

*/

__global__ void cuda_compute_flux(int nelr,int *elements_surrounding_elements,float *normals,float *variables,float *fluxes)

{

  const double smoothing_coefficient = (double )0.2;

  const int i = (blockDim . x * blockIdx . x + threadIdx . x);

  int j;

  int nb;

  float3 normal;

  float norm_len;

  float factor;

  float density_i =( variables[i + 0 * nelr]);

  float3 momentum_i;

  momentum_i . x =( variables[i + (1 + 0) * nelr]);

  momentum_i . y =( variables[i + (1 + 1) * nelr]);

  momentum_i . z =( variables[i + (1 + 2) * nelr]);

  float density_energy_i =( variables[i + (1 + 3) * nelr]);

  float3 velocity_i;

  compute_velocity_1(density_i,momentum_i,velocity_i);

  float speed_sqd_i = compute_speed_sqd_1(velocity_i);

  float speed_i = sqrt(speed_sqd_i);

  float pressure_i = compute_pressure_1(density_i,density_energy_i,speed_sqd_i);

  float speed_of_sound_i = compute_speed_of_sound_1(density_i,pressure_i);

  float3 flux_contribution_i_momentum_x;

  float3 flux_contribution_i_momentum_y;

  float3 flux_contribution_i_momentum_z;

  float3 flux_contribution_i_density_energy;

  compute_flux_contribution(density_i,momentum_i,density_energy_i,pressure_i,velocity_i,flux_contribution_i_momentum_x,flux_contribution_i_momentum_y,flux_contribution_i_momentum_z,flux_contribution_i_density_energy);

  float flux_i_density = (float )0.0;

  float3 flux_i_momentum;

  flux_i_momentum . x = ((float )0.0);

  flux_i_momentum . y = ((float )0.0);

  flux_i_momentum . z = ((float )0.0);

  float flux_i_density_energy = (float )0.0;

  float3 velocity_nb;

  float density_nb;

  float density_energy_nb;

  float3 momentum_nb;

  float3 flux_contribution_nb_momentum_x;

  float3 flux_contribution_nb_momentum_y;

  float3 flux_contribution_nb_momentum_z;

  float3 flux_contribution_nb_density_energy;

  float speed_sqd_nb;

  float speed_of_sound_nb;

  float pressure_nb;

#pragma unroll

  for (j = 0; j < 4; j++) {

    nb = elements_surrounding_elements[i + j * nelr];

    normal . x = normals[i + (j + 0 * 4) * nelr];

    normal . y = normals[i + (j + 1 * 4) * nelr];

    normal . z = normals[i + (j + 2 * 4) * nelr];

    norm_len =( sqrt(normal . x * normal . x + normal . y * normal . y + normal . z * normal . z));

// a legitimate neighbor

    if (nb >= 0) {

      density_nb =( variables[nb + 0 * nelr]);

      momentum_nb . x =( variables[nb + (1 + 0) * nelr]);

      momentum_nb . y =( variables[nb + (1 + 1) * nelr]);

      momentum_nb . z =( variables[nb + (1 + 2) * nelr]);

      density_energy_nb =( variables[nb + (1 + 3) * nelr]);

      compute_velocity_2(density_nb,momentum_nb,velocity_nb);

      speed_sqd_nb = compute_speed_sqd_2(velocity_nb);

      pressure_nb = compute_pressure_2(density_nb,density_energy_nb,speed_sqd_nb);

      speed_of_sound_nb = compute_speed_of_sound_2(density_nb,pressure_nb);

      compute_flux_contribution_1(density_nb,momentum_nb,density_energy_nb,pressure_nb,velocity_nb,flux_contribution_nb_momentum_x,flux_contribution_nb_momentum_y,flux_contribution_nb_momentum_z,flux_contribution_nb_density_energy);

// artificial viscosity

      factor = -norm_len * smoothing_coefficient * ((float )0.5) * (speed_i + sqrt(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);

      flux_i_density =      flux_i_density  +  factor * (density_i - density_nb);

      flux_i_density_energy =      flux_i_density_energy  +  factor * (density_energy_i - density_energy_nb);

      flux_i_momentum . x =      flux_i_momentum . x  +  factor * (momentum_i . x - momentum_nb . x);

      flux_i_momentum . y =      flux_i_momentum . y  +  factor * (momentum_i . y - momentum_nb . y);

      flux_i_momentum . z =      flux_i_momentum . z  +  factor * (momentum_i . z - momentum_nb . z);

// accumulate cell-centered fluxes

      factor =( ((float )0.5) * normal . x);

      flux_i_density =      flux_i_density  +  factor * (momentum_nb . x + momentum_i . x);

      flux_i_density_energy =      flux_i_density_energy  +  factor * (flux_contribution_nb_density_energy . x + flux_contribution_i_density_energy . x);

      flux_i_momentum . x =      flux_i_momentum . x  +  factor * (flux_contribution_nb_momentum_x . x + flux_contribution_i_momentum_x . x);

      flux_i_momentum . y =      flux_i_momentum . y  +  factor * (flux_contribution_nb_momentum_y . x + flux_contribution_i_momentum_y . x);

      flux_i_momentum . z =      flux_i_momentum . z  +  factor * (flux_contribution_nb_momentum_z . x + flux_contribution_i_momentum_z . x);

      factor =( ((float )0.5) * normal . y);

      flux_i_density =      flux_i_density  +  factor * (momentum_nb . y + momentum_i . y);

      flux_i_density_energy =      flux_i_density_energy  +  factor * (flux_contribution_nb_density_energy . y + flux_contribution_i_density_energy . y);

      flux_i_momentum . x =      flux_i_momentum . x  +  factor * (flux_contribution_nb_momentum_x . y + flux_contribution_i_momentum_x . y);

      flux_i_momentum . y =      flux_i_momentum . y  +  factor * (flux_contribution_nb_momentum_y . y + flux_contribution_i_momentum_y . y);

      flux_i_momentum . z =      flux_i_momentum . z  +  factor * (flux_contribution_nb_momentum_z . y + flux_contribution_i_momentum_z . y);

      factor =( ((float )0.5) * normal . z);

      flux_i_density =      flux_i_density  +  factor * (momentum_nb . z + momentum_i . z);

      flux_i_density_energy =      flux_i_density_energy  +  factor * (flux_contribution_nb_density_energy . z + flux_contribution_i_density_energy . z);

      flux_i_momentum . x =      flux_i_momentum . x  +  factor * (flux_contribution_nb_momentum_x . z + flux_contribution_i_momentum_x . z);

      flux_i_momentum . y =      flux_i_momentum . y  +  factor * (flux_contribution_nb_momentum_y . z + flux_contribution_i_momentum_y . z);

      flux_i_momentum . z =      flux_i_momentum . z  +  factor * (flux_contribution_nb_momentum_z . z + flux_contribution_i_momentum_z . z);

    }

     else 

// a wing boundary

if (nb == - 1) {

      flux_i_momentum . x =      flux_i_momentum . x  +  normal . x * pressure_i;

      flux_i_momentum . y =      flux_i_momentum . y  +  normal . y * pressure_i;

      flux_i_momentum . z =      flux_i_momentum . z  +  normal . z * pressure_i;

    }

     else 

// a far field boundary

if (nb == - 2) {

      factor =( ((float )0.5) * normal . x);

      flux_i_density =      flux_i_density  +  factor * (ff_variable[1 + 0] + momentum_i . x);

      flux_i_density_energy =      flux_i_density_energy  +  factor * (ff_flux_contribution_density_energy[0] . x + flux_contribution_i_density_energy . x);

      flux_i_momentum . x =      flux_i_momentum . x  +  factor * (ff_flux_contribution_momentum_x[0] . x + flux_contribution_i_momentum_x . x);

      flux_i_momentum . y =      flux_i_momentum . y  +  factor * (ff_flux_contribution_momentum_y[0] . x + flux_contribution_i_momentum_y . x);

      flux_i_momentum . z =      flux_i_momentum . z  +  factor * (ff_flux_contribution_momentum_z[0] . x + flux_contribution_i_momentum_z . x);

      factor =( ((float )0.5) * normal . y);

      flux_i_density =      flux_i_density  +  factor * (ff_variable[1 + 1] + momentum_i . y);

      flux_i_density_energy =      flux_i_density_energy  +  factor * (ff_flux_contribution_density_energy[0] . y + flux_contribution_i_density_energy . y);

      flux_i_momentum . x =      flux_i_momentum . x  +  factor * (ff_flux_contribution_momentum_x[0] . y + flux_contribution_i_momentum_x . y);

      flux_i_momentum . y =      flux_i_momentum . y  +  factor * (ff_flux_contribution_momentum_y[0] . y + flux_contribution_i_momentum_y . y);

      flux_i_momentum . z =      flux_i_momentum . z  +  factor * (ff_flux_contribution_momentum_z[0] . y + flux_contribution_i_momentum_z . y);

      factor =( ((float )0.5) * normal . z);

      flux_i_density =      flux_i_density  +  factor * (ff_variable[1 + 2] + momentum_i . z);

      flux_i_density_energy =      flux_i_density_energy  +  factor * (ff_flux_contribution_density_energy[0] . z + flux_contribution_i_density_energy . z);

      flux_i_momentum . x =      flux_i_momentum . x  +  factor * (ff_flux_contribution_momentum_x[0] . z + flux_contribution_i_momentum_x . z);

      flux_i_momentum . y =      flux_i_momentum . y  +  factor * (ff_flux_contribution_momentum_y[0] . z + flux_contribution_i_momentum_y . z);

      flux_i_momentum . z =      flux_i_momentum . z  +  factor * (ff_flux_contribution_momentum_z[0] . z + flux_contribution_i_momentum_z . z);

    }

  }

  fluxes[i + 0 * nelr] = flux_i_density;

  fluxes[i + (1 + 0) * nelr] = flux_i_momentum . x;

  fluxes[i + (1 + 1) * nelr] = flux_i_momentum . y;

  fluxes[i + (1 + 2) * nelr] = flux_i_momentum . z;

  fluxes[i + (1 + 3) * nelr] = flux_i_density_energy;

}

void compute_flux(int nelr,int *elements_surrounding_elements,float *normals,float *variables,float *fluxes)

{

  ::dim3 Dg((nelr / 128));

  ::dim3 Db(128);

  cuda_compute_flux<<<Dg,Db>>>(nelr,elements_surrounding_elements,normals,variables,fluxes);

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {

    fprintf(stderr,"GPUassert: %s compute_flux failed\n",(hipGetErrorString(error)));

    exit(- 1);

  }

}

__global__ void cuda_time_step(int j,int nelr,float *old_variables,float *variables,float *step_factors,float *fluxes)

{

  const int i = (blockDim . x * blockIdx . x + threadIdx . x);

  float factor = step_factors[i] / ((float )(3 + 1 - j));

  variables[i + 0 * nelr] = old_variables[i + 0 * nelr] + factor * fluxes[i + 0 * nelr];

  variables[i + (1 + 3) * nelr] = old_variables[i + (1 + 3) * nelr] + factor * fluxes[i + (1 + 3) * nelr];

  variables[i + (1 + 0) * nelr] = old_variables[i + (1 + 0) * nelr] + factor * fluxes[i + (1 + 0) * nelr];

  variables[i + (1 + 1) * nelr] = old_variables[i + (1 + 1) * nelr] + factor * fluxes[i + (1 + 1) * nelr];

  variables[i + (1 + 2) * nelr] = old_variables[i + (1 + 2) * nelr] + factor * fluxes[i + (1 + 2) * nelr];

}

void time_step(int j,int nelr,float *old_variables,float *variables,float *step_factors,float *fluxes)

{

  ::dim3 Dg((nelr / 128));

  ::dim3 Db(128);

  cuda_time_step<<<Dg,Db>>>(j,nelr,old_variables,variables,step_factors,fluxes);

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {

    fprintf(stderr,"GPUassert: %s update failed\n",(hipGetErrorString(error)));

    exit(- 1);

  }

}

/*

 * Main function

 */

int main(int argc,char **argv)

{

  if (argc < 2) {

    (std::cout<<"specify data file name") << endl;

    return 0;

  }

  const char *data_file_name = argv[1];

  struct timeval start_t;

  struct timeval end_t;

  struct timeval skt_t;

  struct timeval ske_t;

  struct hipDeviceProp_t prop;

  int dev;

// CUDA_SAFE_CALL(hipSetDevice(0));

// CUDA_SAFE_CALL(hipGetDevice(&dev));

// CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, dev));

  check((hipSetDevice(0)),"hipSetDevice(0)","euler3d_gdd_real.cu",473);

  check((hipGetDevice(&dev)),"hipGetDevice(&dev)","euler3d_gdd_real.cu",474);

  check((hipGetDeviceProperties(&prop,dev)),"hipGetDeviceProperties(&prop, dev)","euler3d_gdd_real.cu",475);

  printf("Name:                     %s\n",prop . name);

// set far field conditions and load them into constant memory on the gpu

  float h_ff_variable[5];

  const double angle_of_attack = ((double )(3.1415926535897931 / 180.0)) * ((double )0.0);

  h_ff_variable[0] = ((float )1.4);

  float ff_pressure = (float )1.0;

  float ff_speed_of_sound =(float)( sqrt(1.4 * ff_pressure / h_ff_variable[0]));

  float ff_speed = ((float )1.2) * ff_speed_of_sound;

  float3 ff_velocity;

  ff_velocity . x = ff_speed * ((float )(cos((float )angle_of_attack)));

  ff_velocity . y = ff_speed * ((float )(sin((float )angle_of_attack)));

  ff_velocity . z = 0.0;

  h_ff_variable[1 + 0] = h_ff_variable[0] * ff_velocity . x;

  h_ff_variable[1 + 1] = h_ff_variable[0] * ff_velocity . y;

  h_ff_variable[1 + 2] = h_ff_variable[0] * ff_velocity . z;

  h_ff_variable[1 + 3] = h_ff_variable[0] * (((float )0.5) * (ff_speed * ff_speed)) + ff_pressure / ((float )(1.4 - 1.0));

  float3 h_ff_momentum;

  h_ff_momentum . x =(  *(h_ff_variable + 1 + 0));

  h_ff_momentum . y =(  *(h_ff_variable + 1 + 1));

  h_ff_momentum . z =(  *(h_ff_variable + 1 + 2));

  float3 h_ff_flux_contribution_momentum_x;

  float3 h_ff_flux_contribution_momentum_y;

  float3 h_ff_flux_contribution_momentum_z;

  float3 h_ff_flux_contribution_density_energy;

  compute_flux_contribution_host(h_ff_variable[0],h_ff_momentum,h_ff_variable[1 + 3],ff_pressure,ff_velocity,h_ff_flux_contribution_momentum_x,h_ff_flux_contribution_momentum_y,h_ff_flux_contribution_momentum_z,h_ff_flux_contribution_density_energy);

// copy far field conditions to the gpu

  check((hipMemcpyToSymbol(HIP_SYMBOL(ff_variable),h_ff_variable,(1 + 3 + 1) * sizeof(float ))),"hipMemcpyToSymbol(HIP_SYMBOL(ff_variable), h_ff_variable, NVAR*sizeof(float))","euler3d_float.cu",512);

  check((hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_x),(&h_ff_flux_contribution_momentum_x),sizeof(::float3 ))),"hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_x), &h_ff_flux_contribution_momentum_x, sizeof(float3))","euler3d_float.cu",513);

  check((hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_y),(&h_ff_flux_contribution_momentum_y),sizeof(::float3 ))),"hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_y), &h_ff_flux_contribution_momentum_y, sizeof(float3))","euler3d_float.cu",514);

  check((hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_z),(&h_ff_flux_contribution_momentum_z),sizeof(::float3 ))),"hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_z), &h_ff_flux_contribution_momentum_z, sizeof(float3))","euler3d_float.cu",515);

  check((hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_density_energy),(&h_ff_flux_contribution_density_energy),sizeof(::float3 ))),"hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_density_energy), &h_ff_flux_contribution_density_energy, sizeof(float3))","euler3d_float.cu",517);

  int nel;

  int nelr;

// read in domain geometry

  float *areas;

  int *elements_surrounding_elements;

  float *normals;

  std::ifstream file(data_file_name);

  file >> nel;

  nelr = 128 * (nel / 128 + min(1,nel % 128));

  float *h_areas = new float [nelr];

  int *h_elements_surrounding_elements = new int [nelr * 4];

  float *h_normals = new float [nelr * 3 * 4];

  double input;

// read in data

  for (int i = 0; i < nel; i++) {

    file >> input;

    h_areas[i] = input;

    for (int j = 0; j < 4; j++) {

      file >> input;

      h_elements_surrounding_elements[i + j * nelr] = input;

      if (h_elements_surrounding_elements[i + j * nelr] < 0) 

        h_elements_surrounding_elements[i + j * nelr] = - 1;

//it's coming in with Fortran numbering				

      h_elements_surrounding_elements[i + j * nelr]--;

      for (int k = 0; k < 3; k++) {

        file >> input;

        h_normals[i + (j + k * 4) * nelr] = input;

        h_normals[i + (j + k * 4) * nelr] = -h_normals[i + (j + k * 4) * nelr];

      }

    }

  }

// fill in remaining data

  int last = nel - 1;

  for (int i = nel; i < nelr; i++) {

    h_areas[i] = h_areas[last];

    for (int j = 0; j < 4; j++) {

// duplicate the last element

      h_elements_surrounding_elements[i + j * nelr] = h_elements_surrounding_elements[last + j * nelr];

      for (int k = 0; k < 3; k++) 

        h_normals[last + (j + k * 4) * nelr] = h_normals[last + (j + k * 4) * nelr];

    }

  }

  gettimeofday(&start_t,0L);

  areas = alloc< float  > (nelr);

  upload< float  > (areas,h_areas,nelr);

  elements_surrounding_elements = alloc< int  > (nelr * 4);

  upload< int  > (elements_surrounding_elements,h_elements_surrounding_elements,nelr * 4);

  normals = alloc< float  > (nelr * 3 * 4);

  upload< float  > (normals,h_normals,nelr * 3 * 4);

  delete []h_areas;

  delete []h_elements_surrounding_elements;

  delete []h_normals;

// Create arrays and set initial conditions

  float *variables = alloc< float  > (nelr * (1 + 3 + 1));

  initialize_variables(nelr,variables);

  float *old_variables = alloc< float  > (nelr * (1 + 3 + 1));

  float *fluxes = alloc< float  > (nelr * (1 + 3 + 1));

  float *step_factors = alloc< float  > (nelr);

// make sure all memory is gdd_really allocated before we start timing

  initialize_variables_1(nelr,old_variables);

  initialize_variables_2(nelr,fluxes);

  hipMemset((void *)step_factors,0,sizeof(float ) * nelr);

// make sure CUDA isn't still doing something before we start timing

  hipDeviceSynchronize();

// these need to be computed the first time in order to compute time step

  (std::cout<<"Starting...") << endl;

  hipError_t error;

//StopWatchInterface *timer = NULL;

//sdkCreateTimer( &timer);

//sdkStartTimer( &timer);

// Begin iterations

  gettimeofday(&skt_t,0L);

  for (int i = 0; i < 2000; i++) {

    copy< float  > (old_variables,variables,nelr * (1 + 3 + 1));

// for the first iteration we compute the time step

    compute_step_factor(nelr,variables,areas,step_factors);

    error = hipGetLastError();

    if (error != hipSuccess) {

      fprintf(stderr,"GPUassert: %s compute_step_factor failed\n",(hipGetErrorString(error)));

      exit(- 1);

    }

    for (int j = 0; j < 3; j++) {

      compute_flux(nelr,elements_surrounding_elements,normals,variables,fluxes);

      error = hipGetLastError();

      if (error != hipSuccess) {

        fprintf(stderr,"GPUassert: %s compute_flux failed\n",(hipGetErrorString(error)));

        exit(- 1);

      }

      time_step(j,nelr,old_variables,variables,step_factors,fluxes);

      error = hipGetLastError();

      if (error != hipSuccess) {

        fprintf(stderr,"GPUassert: %s time_step failed\n",(hipGetErrorString(error)));

        exit(- 1);

      }

    }

  }

  hipDeviceSynchronize();

  gettimeofday(&ske_t,0L);

//sdkStopTimer(&timer);  

  gettimeofday(&end_t,0L);
  float* d_variables = new float[nelr*NVAR];
	download(d_variables, variables, nelr*NVAR);
  mpf_t val_x, val_y, val_in, err;
  mpf_init2(val_x, 128);
  mpf_init2(val_y, 128);
  mpf_init2(val_in, 128);
  mpf_init2(err, 128);
  FILE* infile = fopen("density_ref.txt", "r");
  for(int i = 0; i < nel; i++) {
    gmp_fscanf(infile, "%Fe\n", val_in);
    mpf_set_d(val_x, d_variables[i + VAR_DENSITY_ENERGY*nelr]);
    mpf_sub(val_x, val_x, val_in);
    mpf_abs(val_y, val_x);
    mpf_div(val_x, val_y, val_in);
    if (i==0)
      mpf_set(err, val_x);
    else
      mpf_add(err, err, val_x);
  }
  mpf_div_ui(err, err, nel);
  fclose(infile);
  gmp_printf("error: %10.5Fe\n", err);

  ((std::cout<<"time: ") << ((end_t . tv_sec - start_t . tv_sec) + (end_t . tv_usec - start_t . tv_usec) * 1e-6)) << endl;

  ((std::cout<<"kernel: ") << ((ske_t . tv_sec - skt_t . tv_sec) + (ske_t . tv_usec - skt_t . tv_usec) * 1e-6)) << endl;

//std::cout  << (sdkGetAverageTimerValue(&timer)/1000.0)  / iterations << " seconds per iteration" << std::endl;

//std::cout << "Saving solution..." << std::endl;

//dump(variables, nel, nelr);

//std::cout << "Saved solution..." << std::endl;

//

//std::cout << "Cleaning up..." << std::endl;

  dealloc< float  > (areas);

  dealloc< int  > (elements_surrounding_elements);

  dealloc< float  > (normals);

  dealloc< float  > (variables);

  dealloc< float  > (old_variables);

  dealloc< float  > (fluxes);

  dealloc< float  > (step_factors);

  (std::cout<<"Done...") << endl;

  return 0;

}

