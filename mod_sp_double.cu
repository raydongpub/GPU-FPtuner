
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>
#define NPB_VERSION "3.3.1"

using namespace std;


#define min(x,y) (x) <= (y) ? (x) : (y)
#define max(x,y) (x) >= (y) ? (x) : (y)

// block sizes for CUDA kernels
#define NORM_BLOCK 32
#define SOLVE_BLOCK 32
#define ERHS_BLOCK 32

// timer constants
#define t_total 0
#define t_rhsx 1
#define t_rhsy 2
#define t_rhsz 3
#define t_rhs 4
#define t_xsolve 5
#define t_ysolve 6
#define t_zsolve 7
#define t_rdis1 8
#define t_rdis2 9
#define t_txinvr 10
#define t_pinvr 11
#define t_ninvr 12
#define t_tzetar 13
#define t_add 14
#define t_last 15


//---------------------------------------------------------------------
// diffusion coefficients
//---------------------------------------------------------------------
#define dx1 0.75
#define dx2 0.75
#define dx3 0.75
#define dx4 0.75
#define dx5 0.75
#define dy1 0.75
#define dy2 0.75
#define dy3 0.75
#define	dy4 0.75
#define dy5 0.75
#define dz1 1.0
#define dz2 1.0
#define dz3 1.0
#define dz4 1.0
#define dz5 1.0
//#define dxmax max(dx3,dx4)
//#define dymax max(dy2,dy4)
//#define dzmax max(dz2,dz3)
#define dxmax dx3
#define dymax dy2
#define dzmax dz2
//---------------------------------------------------------------------
//   fourth difference dissipation
//---------------------------------------------------------------------
#define dssp (max(max(dx1,dy1),dz1)*.25)
#define c4dssp (4.0*dssp)
#define c5dssp (5.0*dssp)

#define c1 1.4
#define c2 0.4
#define	c3 0.1
#define c4 1.0
#define c5 1.4
#define c1c2 (c1*c2)
#define c1c5 (c1*c5)
#define c3c4 (c3*c4)
#define c1345 (c1c5*c3c4)
#define conz1 (1.0-c1c5)
#define c2iv 2.5
#define con43 (4.0/3.0)
#define con16 (1.0/6.0)

// macros to linearize multidimensional array accesses 
#define u(m,i,j,k) u[(i)+nx*((j)+ny*((k)+nz*(m)))]
#define forcing(m,i,j,k) forcing[(i)+nx*((j)+ny*((k)+nz*(m)))]
#define rhs(m,i,j,k) rhs[m+(i)*5+(j)*5*nx+(k)*5*nx*ny]
#define rho_i(i,j,k) rho_i[i+(j)*nx+(k)*nx*ny]
#define us(i,j,k) us[i+(j)*nx+(k)*nx*ny]
#define vs(i,j,k) vs[i+(j)*nx+(k)*nx*ny]
#define ws(i,j,k) ws[i+(j)*nx+(k)*nx*ny]
#define square(i,j,k) square[i+(j)*nx+(k)*nx*ny]
#define qs(i,j,k) qs[i+(j)*nx+(k)*nx*ny]
#define speed(i,j,k) speed[i+(j)*nx+(k)*nx*ny]

static void inline HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__constant__ double tx1, tx2, tx3, ty1, ty2, ty3, tz1, tz2, tz3;
__constant__ double bt, dt, dtdssp;
__constant__ double dnxm1, dnym1, dnzm1;
__constant__ double dttx1, dttx2, dtty1, dtty2, dttz1, dttz2, c2dttx1, c2dtty1, c2dttz1;
__constant__ double comz1, comz4, comz5, comz6, c3c4tx3, c3c4ty3, c3c4tz3;
__constant__ double xxcon1, xxcon2, xxcon3, xxcon4, xxcon5, dx1tx1, dx2tx1, dx3tx1, dx4tx1, dx5tx1;
__constant__ double yycon1, yycon2, yycon3, yycon4, yycon5, dy1ty1, dy2ty1, dy3ty1, dy4ty1, dy5ty1;
__constant__ double zzcon1, zzcon2, zzcon3, zzcon4, zzcon5, dz1tz1, dz2tz1, dz3tz1, dz4tz1, dz5tz1;
__constant__ double ce[13][5];


//---------------------------------------------------------------------
// exact_rhs computation
//---------------------------------------------------------------------
__device__ static void exact_solution_kernel (const double xi, const double eta, const double zeta, double *dtemp) {
	for (int m = 0; m < 5; m++)
		dtemp[m] = ce[0][m] + xi*(ce[1][m] + xi*(ce[4][m] + xi*(ce[7][m] + xi*ce[10][m]))) +
				eta*(ce[2][m] + eta*(ce[5][m] + eta*(ce[8][m] + eta*ce[11][m])))+
				zeta*(ce[3][m] + zeta*(ce[6][m] + zeta*(ce[9][m] + zeta*ce[12][m])));
}

__global__ static void exact_rhs_kernel_init (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	k = blockIdx.y;
	j = blockIdx.x;
	i = threadIdx.x;
	for (m = 0; m < 5; m++) forcing(m,i,j,k) = (double)0.0;
}

__global__ static void exact_rhs_kernel_x (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtemp[5], dtpp;
	double ue[5][5], buf[3][5], cuf[3], q[3];

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	j = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || j >= ny-1) return;

	zeta = (double)k * dnzm1;
	eta = (double)j * dnym1;
	//---------------------------------------------------------------------
	//      xi-direction flux differences                      
	//---------------------------------------------------------------------
	for (i = 0; i < 3; i++) {
		xi = (double)i * dnxm1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[i+1][m] = dtemp[m];
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[i][m] = dtpp*dtemp[m];
		cuf[i] = buf[i][1] * buf[i][1];
		buf[i][0] = cuf[i] + buf[i][2] * buf[i][2] + buf[i][3] * buf[i][3];
		q[i] = 0.5 * (buf[i][1]*ue[i+1][1] + buf[i][2]*ue[i+1][2] + buf[i][3]*ue[i+1][3]);
	}
	for (i = 1; i < nx-1; i++) {
		if (i+2 < nx) {
			xi = (double)(i+2) * dnxm1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}
		
		dtemp[0] = 0.0 - tx2*(ue[3][1]-ue[1][1])+ dx1tx1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = 0.0 - tx2*((ue[3][1]*buf[2][1]+c2*(ue[3][4]-q[2]))-(ue[1][1]*buf[0][1]+c2*(ue[1][4]-q[0])))+xxcon1*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dx2tx1*(ue[3][1]-2.0*ue[2][1]+ue[1][1]);
		dtemp[2] = 0.0 - tx2*(ue[3][2]*buf[2][1]-ue[1][2]*buf[0][1])+xxcon2*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dx3tx1*(ue[3][2]-2.0*ue[2][2]+ue[1][2]);
		dtemp[3] = 0.0 - tx2*(ue[3][3]*buf[2][1]-ue[1][3]*buf[0][1])+xxcon2*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dx4tx1*(ue[3][3]-2.0*ue[2][3]+ue[1][3]);
		dtemp[4] = 0.0 - tx2*(buf[2][1]*(c1*ue[3][4]-c2*q[2])-buf[0][1]*(c1*ue[1][4]-c2*q[0]))+0.5*xxcon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+xxcon4*(cuf[2]-2.0*cuf[1]+cuf[0])+
					xxcon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dx5tx1*(ue[3][4]-2.0*ue[2][4]+ ue[1][4]);
		//---------------------------------------------------------------------
		//            Fourth-order dissipation                         
		//---------------------------------------------------------------------
		if (i == 1) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(5.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i == 2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i >= 3 && i < nx-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m]+6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i == nx-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] +6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (i == nx-2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] + 5.0*ue[2][m]);
		}

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (i < nx-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp*ue[3][m];
			cuf[2] = buf[2][1] * buf[2][1];
			buf[2][0] = cuf[2] + buf[2][2] * buf[2][2] + buf[2][3] * buf[2][3];
			q[2] = 0.5 * (buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

__global__ static void exact_rhs_kernel_y (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtemp[5], dtpp;
	double ue[5][5], buf[3][5], cuf[3], q[3];

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || i >= nx-1) return;

	zeta = (double)k * dnzm1;
	xi = (double)i * dnxm1;
	//---------------------------------------------------------------------
	//  eta-direction flux differences             
	//---------------------------------------------------------------------
	for (j = 0; j < 3; j++) {
		eta = (double)j * dnym1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[j+1][m] = dtemp[m];
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[j][m] = dtpp * dtemp[m];
		cuf[j] = buf[j][2] * buf[j][2];
		buf[j][0] = cuf[j] + buf[j][1] * buf[j][1] + buf[j][3] * buf[j][3];
		q[j] = 0.5*(buf[j][1]*ue[j+1][1] + buf[j][2]*ue[j+1][2] + buf[j][3]*ue[j+1][3]);
	}

	for (j = 1; j < ny-1; j++) {
		if (j+2 < ny) {
			eta = (double)(j+2) * dnym1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}

		dtemp[0] = forcing(0,i,j,k) - ty2*(ue[3][2]-ue[1][2])+ dy1ty1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = forcing(1,i,j,k) - ty2*(ue[3][1]*buf[2][2]-ue[1][1]*buf[0][2])+yycon2*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dy2ty1*(ue[3][1]-2.0*ue[2][1]+ ue[1][1]);
		dtemp[2] = forcing(2,i,j,k) - ty2*((ue[3][2]*buf[2][2]+c2*(ue[3][4]-q[2]))-(ue[1][2]*buf[0][2]+c2*(ue[1][4]-q[0])))+yycon1*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dy3ty1*( ue[3][2]-2.0*ue[2][2] +ue[1][2]);
		dtemp[3] = forcing(3,i,j,k) - ty2*(ue[3][3]*buf[2][2]-ue[1][3]*buf[0][2])+yycon2*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dy4ty1*( ue[3][3]-2.0*ue[2][3]+ ue[1][3]);
		dtemp[4] = forcing(4,i,j,k) - ty2*(buf[2][2]*(c1*ue[3][4]-c2*q[2])-buf[0][2]*(c1*ue[1][4]-c2*q[0]))+0.5*yycon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+yycon4*(cuf[2]-2.0*cuf[1]+cuf[0])+
					yycon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dy5ty1*(ue[3][4]-2.0*ue[2][4]+ue[1][4]);
		//---------------------------------------------------------------------
		//            Fourth-order dissipation                      
		//---------------------------------------------------------------------
		if (j == 1) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (5.0*ue[2][m] - 4.0*ue[3][m] +ue[4][m]);
		} else if (j == 2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] +       ue[4][m]);
		} else if (j >= 3 && j < ny-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (j == ny-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (ue[0][m] - 4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (j == ny-2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (ue[0][m] - 4.0*ue[1][m] + 5.0*ue[2][m]);
		}

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (j < ny-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp * ue[3][m];
			cuf[2] = buf[2][2] * buf[2][2];
			buf[2][0] = cuf[2] + buf[2][1] * buf[2][1] + buf[2][3] * buf[2][3];
			q[2] = 0.5*(buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

__global__ static void exact_rhs_kernel_z (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtpp, dtemp[5];
	double ue[5][5], buf[3][5], cuf[3], q[3];

	j = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (j >= ny-1 || i >= nx-1) return;

	eta = (double)j * dnym1;
	xi = (double)i * dnxm1;
	//---------------------------------------------------------------------
	//      zeta-direction flux differences                      
	//---------------------------------------------------------------------
	for (k = 0; k < 3; k++) {
		zeta = (double)k * dnzm1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[k+1][m] = dtemp[m];
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[k][m] = dtpp * dtemp[m];
		cuf[k] = buf[k][3] * buf[k][3];
		buf[k][0] = cuf[k] + buf[k][1] * buf[k][1] + buf[k][2] * buf[k][2];
		q[k] = 0.5*(buf[k][1]*ue[k+1][1] + buf[k][2]*ue[k+1][2] + buf[k][3]*ue[k+1][3]);
	}

	for (k = 1; k < nz-1; k++) {
		if (k+2 < nz) {
			zeta = (double)(k+2) * dnzm1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}

		dtemp[0] = forcing(0,i,j,k) - tz2*(ue[3][3]-ue[1][3])+dz1tz1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = forcing(1,i,j,k) - tz2*(ue[3][1]*buf[2][3]-ue[1][1]*buf[0][3])+zzcon2*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dz2tz1*(ue[3][1]-2.0*ue[2][1]+ue[1][1]);
		dtemp[2] = forcing(2,i,j,k) - tz2*(ue[3][2]*buf[2][3]-ue[1][2]*buf[0][3])+zzcon2*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dz3tz1*(ue[3][2]-2.0*ue[2][2]+ue[1][2]);
		dtemp[3] = forcing(3,i,j,k) - tz2*((ue[3][3]*buf[2][3]+c2*(ue[3][4]-q[2]))-(ue[1][3]*buf[0][3]+c2*(ue[1][4]-q[0])))+zzcon1*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dz4tz1*(ue[3][3]-2.0*ue[2][3] +ue[1][3]);
		dtemp[4] = forcing(4,i,j,k) - tz2*(buf[2][3]*(c1*ue[3][4]-c2*q[2])-buf[0][3]*(c1*ue[1][4]-c2*q[0]))+0.5*zzcon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+
					zzcon4*(cuf[2]-2.0*cuf[1]+cuf[0])+zzcon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dz5tz1*(ue[3][4]-2.0*ue[2][4]+ue[1][4]);
		//---------------------------------------------------------------------
		//            Fourth-order dissipation
		//---------------------------------------------------------------------
		if (k == 1) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(5.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k == 2) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(-4.0*ue[1][m]+6.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k >= 3 && k < nz-3) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m]+6.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k == nz-3) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (k == nz-2) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m]+5.0*ue[2][m]);
		}
		//---------------------------------------------------------------------
		// now change the sign of the forcing function, 
		//---------------------------------------------------------------------
		for (m = 0; m < 5; m++) forcing(m,i,j,k) = -1.0 * dtemp[m];

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (k < nz-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp * ue[3][m];
			cuf[2] = buf[2][3] * buf[2][3];
			buf[2][0] = cuf[2] + buf[2][1] * buf[2][1] + buf[2][2] * buf[2][2];
			q[2] = 0.5*(buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

void exact_rhs (double* forcing, int nx, int ny, int nz) {
	dim3 gridinit(ny,nz);
	exact_rhs_kernel_init<<<gridinit,nx>>>(forcing, nx, ny, nz);

	int yblock = min(ERHS_BLOCK,ny);
	int ygrid = (ny+yblock-1)/yblock;
	int zblock_y = min(ERHS_BLOCK/yblock,nz);
	int zgrid_y = (nz+zblock_y-1)/zblock_y;
	dim3 grid_x(zgrid_y,ygrid), block_x(zblock_y,yblock);
	exact_rhs_kernel_x<<<grid_x,block_x>>>(forcing, nx, ny, nz);

	int xblock = min(ERHS_BLOCK,nx);
	int xgrid = (nx+xblock-1)/xblock;
	int zblock_x = min(ERHS_BLOCK/xblock,nz);
	int zgrid_x = (nz+zblock_x-1)/zblock_x;
	dim3 grid_y(zgrid_x,xgrid), block_y(zblock_x,xblock);
	exact_rhs_kernel_y<<<grid_y,block_y>>>(forcing, nx, ny, nz);

	int yblock_x = min(ERHS_BLOCK/xblock,ny);
	int ygrid_x = (ny+yblock_x-1)/yblock_x;
	dim3 grid_z(ygrid_x,xgrid), block_z(yblock_x,xblock);
	exact_rhs_kernel_z<<<grid_z,block_z>>>(forcing, nx, ny, nz);
}


//---------------------------------------------------------------------
// initialize_kernel
//---------------------------------------------------------------------
__global__ static void initialize_kernel (double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, temp[5];
	double Pface11[5], Pface12[5], Pface21[5], Pface22[5], Pface31[5], Pface32[5];
  double zero, one;

	k = blockIdx.x;
	j = blockIdx.y;
	i = threadIdx.x;

	//---------------------------------------------------------------------
	//  Later (in compute_rhs) we compute 1/u for every element. A few of 
	//  the corner elements are not used, but it convenient (and faster) 
	//  to compute the whole thing with a simple loop. Make sure those 
	//  values are nonzero by initializing the whole thing here. 
	//---------------------------------------------------------------------
	u(0,i,j,k) = (double)1.0;
	u(1,i,j,k) = (double)0.0;
	u(2,i,j,k) = (double)0.0;
	u(3,i,j,k) = (double)0.0;
	u(4,i,j,k) = (double)1.0;
  zero = (double)0.0;
  one = (double)1.0;

	//---------------------------------------------------------------------
	// first store the "interpolated" values everywhere on the zone    
	//---------------------------------------------------------------------
	zeta = (double)k * dnzm1;
	eta = (double)j * dnym1;
	xi = (double)i * dnxm1;
	exact_solution_kernel (zero, eta, zeta, Pface11);
	exact_solution_kernel (one, eta, zeta, Pface12);
	exact_solution_kernel (xi, zero, zeta, Pface21);
	exact_solution_kernel (xi, one, zeta, Pface22);
	exact_solution_kernel (xi, eta, zero, Pface31);
	exact_solution_kernel (xi, eta, one, Pface32);
	for (m = 0; m < 5; m++) {
		double Pxi = xi * Pface12[m] + (1.0-xi)*Pface11[m];
		double Peta = eta * Pface22[m] + (1.0-eta)*Pface21[m];
		double Pzeta = zeta * Pface32[m] + (1.0-zeta)*Pface31[m];
		u(m,i,j,k) = Pxi + Peta + Pzeta - Pxi*Peta - Pxi*Pzeta - Peta*Pzeta + Pxi*Peta*Pzeta;
	}

	//---------------------------------------------------------------------
	// now store the exact values on the boundaries        
	//---------------------------------------------------------------------

	//---------------------------------------------------------------------
	// west face                                                  
	//---------------------------------------------------------------------
	xi = (double)0.0;
	if (i == 0) {
		zeta = (double)k * dnzm1;
		eta = (double)j * dnym1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// east face                                                      
	//---------------------------------------------------------------------
	xi = (double)1.0;
	if (i == nx-1) {
		zeta = (double)k * dnzm1;
		eta = (double)j * dnym1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// south face                                                 
	//---------------------------------------------------------------------
	eta = (double)0.0;
	if (j == 0) {
		zeta = (double)k * dnzm1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi,eta,zeta,temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// north face                                    
	//---------------------------------------------------------------------
	eta = (double)1.0;
	if (j == ny-1) {
		zeta = (double)k * dnzm1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi,eta,zeta,temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// bottom face                                       
	//---------------------------------------------------------------------
	zeta = (double)0.0;
	if (k == 0) {
		eta = (double)j * dnym1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// top face     
	//---------------------------------------------------------------------
	zeta = (double)1.0;
	if (k == nz-1) {
		eta = (double)j * dnym1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
}

//---------------------------------------------------------------------
// adi: compute_rhs
//---------------------------------------------------------------------
__global__ static void compute_rhs_kernel_1 (double *rho_i, double *us, double *vs, double *ws, double *speed, double *qs, double *square, const double *u, const int nx, const int ny, const int nz) {
	int i, j, k;
	k = blockIdx.y;
	j = blockIdx.x;
	i = threadIdx.x;
	//---------------------------------------------------------------------
	//      compute the reciprocal of density, and the kinetic energy, 
	//      and the speed of sound. 
	//---------------------------------------------------------------------
	double rho_inv = 1.0/u(0,i,j,k);
	double square_ijk;
	rho_i(i,j,k) = rho_inv;
	us(i,j,k) = u(1,i,j,k) * rho_inv;
	vs(i,j,k) = u(2,i,j,k) * rho_inv;
	ws(i,j,k) = u(3,i,j,k) * rho_inv;
	square(i,j,k) = square_ijk = 0.5*(u(1,i,j,k)*u(1,i,j,k) + u(2,i,j,k)*u(2,i,j,k) + u(3,i,j,k)*u(3,i,j,k)) * rho_inv;
	qs(i,j,k) = square_ijk * rho_inv;
	//---------------------------------------------------------------------
	//               (don't need speed and ainx until the lhs computation)
	//---------------------------------------------------------------------
	speed(i,j,k) = sqrt(c1c2*rho_inv*(u(4,i,j,k) - square_ijk));
}

__global__ static void compute_rhs_kernel_2 (const double *rho_i, const double *us, const double *vs, const double *ws, const double *qs, const double *square, double *rhs, const double *forcing, const double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	k = blockIdx.y;
	j = blockIdx.x;
	i = threadIdx.x;
	double rtmp[5];

	//---------------------------------------------------------------------
	// copy the exact forcing term to the right hand side;  because 
	// this forcing term is known, we can store it on the whole zone
	// including the boundary                   
	//---------------------------------------------------------------------
	for (m = 0; m < 5; m++) rtmp[m] = forcing(m,i,j,k);
	
	//---------------------------------------------------------------------
	//      compute xi-direction fluxes 
	//---------------------------------------------------------------------
	if (k >= 1 && k < nz-1 && j >= 1 && j < ny-1 && i >= 1 && i < nx-1) {
		double uijk = us(i,j,k);
		double up1 = us(i+1,j,k);
		double um1 = us(i-1,j,k);
				
		rtmp[0] = rtmp[0] + dx1tx1*(u(0,i+1,j,k) - 2.0*u(0,i,j,k) + u(0,i-1,j,k)) - tx2*(u(1,i+1,j,k)-u(1,i-1,j,k));
		rtmp[1] = rtmp[1] + dx2tx1*(u(1,i+1,j,k) - 2.0*u(1,i,j,k) + u(1,i-1,j,k)) + xxcon2*con43*(up1-2.0*uijk+um1) - tx2*(u(1,i+1,j,k)*up1 - u(1,i-1,j,k)*um1 + (u(4,i+1,j,k)-square(i+1,j,k)-u(4,i-1,j,k)+square(i-1,j,k))*c2);
		rtmp[2] = rtmp[2] + dx3tx1*(u(2,i+1,j,k) - 2.0*u(2,i,j,k) + u(2,i-1,j,k)) + xxcon2*(vs(i+1,j,k)-2.0*vs(i,j,k)+vs(i-1,j,k)) - tx2*(u(2,i+1,j,k)*up1 - u(2,i-1,j,k)*um1);
		rtmp[3] = rtmp[3] + dx4tx1*(u(3,i+1,j,k) - 2.0*u(3,i,j,k) + u(3,i-1,j,k)) + xxcon2*(ws(i+1,j,k)-2.0*ws(i,j,k)+ws(i-1,j,k)) - tx2*(u(3,i+1,j,k)*up1 - u(3,i-1,j,k)*um1);
		rtmp[4] = rtmp[4] + dx5tx1*(u(4,i+1,j,k) - 2.0*u(4,i,j,k) + u(4,i-1,j,k)) + xxcon3*(qs(i+1,j,k)-2.0*qs(i,j,k)+qs(i-1,j,k))+ xxcon4*(up1*up1-2.0*uijk*uijk+um1*um1) +
				xxcon5*(u(4,i+1,j,k)*rho_i(i+1,j,k) - 2.0*u(4,i,j,k)*rho_i(i,j,k) + u(4,i-1,j,k)*rho_i(i-1,j,k)) - tx2*((c1*u(4,i+1,j,k) - c2*square(i+1,j,k))*up1 - (c1*u(4,i-1,j,k) - c2*square(i-1,j,k))*um1 );
		//---------------------------------------------------------------------
		//      add fourth order xi-direction dissipation               
		//---------------------------------------------------------------------
		if (i == 1) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (5.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
		} else if (i == 2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
		} else if (i >= 3 && i < nx-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * ( u(m,i-2,j,k)-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
		} else if (i == nx-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (u(m,i-2,j,k)-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k) );
		} else if (i == nx-2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (u(m,i-2,j,k)-4.0*u(m,i-1,j,k) + 5.0*u(m,i,j,k));
		}
		//---------------------------------------------------------------------
		//      compute eta-direction fluxes 
		//---------------------------------------------------------------------
		double vijk = vs(i,j,k);
		double vp1 = vs(i,j+1,k);
		double vm1 = vs(i,j-1,k);
		rtmp[0] = rtmp[0] + dy1ty1*(u(0,i,j+1,k) - 2.0*u(0,i,j,k) + u(0,i,j-1,k)) - ty2*(u(2,i,j+1,k)-u(2,i,j-1,k));
		rtmp[1] = rtmp[1] + dy2ty1*(u(1,i,j+1,k) - 2.0*u(1,i,j,k) + u(1,i,j-1,k)) + yycon2*(us(i,j+1,k)-2.0*us(i,j,k)+us(i,j-1,k)) - ty2*(u(1,i,j+1,k)*vp1-u(1,i,j-1,k)*vm1);
		rtmp[2] = rtmp[2] + dy3ty1*(u(2,i,j+1,k) - 2.0*u(2,i,j,k) + u(2,i,j-1,k)) + yycon2*con43*(vp1-2.0*vijk+vm1) - ty2*(u(2,i,j+1,k)*vp1-u(2,i,j-1,k)*vm1+(u(4,i,j+1,k)-square(i,j+1,k)-u(4,i,j-1,k)+square(i,j-1,k))*c2);
		rtmp[3] = rtmp[3] + dy4ty1*(u(3,i,j+1,k) - 2.0*u(3,i,j,k) + u(3,i,j-1,k)) + yycon2*(ws(i,j+1,k)-2.0*ws(i,j,k)+ws(i,j-1,k))-ty2*(u(3,i,j+1,k)*vp1-u(3,i,j-1,k)*vm1);
		rtmp[4] = rtmp[4] + dy5ty1*(u(4,i,j+1,k) - 2.0*u(4,i,j,k) + u(4,i,j-1,k)) + yycon3*(qs(i,j+1,k)-2.0*qs(i,j,k)+qs(i,j-1,k)) + yycon4*(vp1*vp1-2.0*vijk*vijk+vm1*vm1) +
				yycon5*(u(4,i,j+1,k)*rho_i(i,j+1,k)-2.0*u(4,i,j,k)*rho_i(i,j,k)+u(4,i,j-1,k)*rho_i(i,j-1,k)) - ty2*((c1*u(4,i,j+1,k)-c2*square(i,j+1,k))*vp1 - (c1*u(4,i,j-1,k)-c2*square(i,j-1,k))*vm1);
		//---------------------------------------------------------------------
		//      add fourth order eta-direction dissipation         
		//---------------------------------------------------------------------
		if (j == 1) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(5.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
		} else if (j == 2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
		} else if (j >= 3 && j < ny-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
		} else if (j == ny-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k));
		} else if (j == ny-2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+5.0*u(m,i,j,k));
		}
		//---------------------------------------------------------------------
		//      compute zeta-direction fluxes 
		//---------------------------------------------------------------------
		double wijk = ws(i,j,k);
		double wp1 = ws(i,j,k+1);
		double wm1 = ws(i,j,k-1);

		rtmp[0] = rtmp[0] + dz1tz1*(u(0,i,j,k+1)-2.0*u(0,i,j,k)+u(0,i,j,k-1)) - tz2*(u(3,i,j,k+1)-u(3,i,j,k-1));
		rtmp[1] = rtmp[1] + dz2tz1*(u(1,i,j,k+1)-2.0*u(1,i,j,k)+u(1,i,j,k-1)) + zzcon2*(us(i,j,k+1)-2.0*us(i,j,k)+us(i,j,k-1)) - tz2*(u(1,i,j,k+1)*wp1-u(1,i,j,k-1)*wm1);
		rtmp[2] = rtmp[2] + dz3tz1*(u(2,i,j,k+1)-2.0*u(2,i,j,k)+u(2,i,j,k-1)) + zzcon2*(vs(i,j,k+1)-2.0*vs(i,j,k)+vs(i,j,k-1)) - tz2*(u(2,i,j,k+1)*wp1-u(2,i,j,k-1)*wm1);
		rtmp[3] = rtmp[3] + dz4tz1*(u(3,i,j,k+1)-2.0*u(3,i,j,k)+u(3,i,j,k-1)) + zzcon2*con43*(wp1-2.0*wijk+wm1) - tz2*(u(3,i,j,k+1)*wp1-u(3,i,j,k-1)*wm1+(u(4,i,j,k+1)-square(i,j,k+1)-u(4,i,j,k-1)+square(i,j,k-1))*c2);
		rtmp[4] = rtmp[4] + dz5tz1*(u(4,i,j,k+1)-2.0*u(4,i,j,k)+u(4,i,j,k-1)) + zzcon3*(qs(i,j,k+1)-2.0*qs(i,j,k)+qs(i,j,k-1)) + zzcon4*(wp1*wp1-2.0*wijk*wijk+wm1*wm1) +
			zzcon5*(u(4,i,j,k+1)*rho_i(i,j,k+1)-2.0*u(4,i,j,k)*rho_i(i,j,k)+u(4,i,j,k-1)*rho_i(i,j,k-1)) - tz2*((c1*u(4,i,j,k+1)-c2*square(i,j,k+1))*wp1-(c1*u(4,i,j,k-1)-c2*square(i,j,k-1))*wm1);
		//---------------------------------------------------------------------
		//      add fourth order zeta-direction dissipation                
		//---------------------------------------------------------------------
		if (k == 1) {
			for (m = 0; m < 5; m++)	rtmp[m] = rtmp[m] - dssp*(5.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
		} else if (k == 2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
		} else if (k >= 3 && k < nz-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
		} else if (k == nz-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1));
		} else if (k == nz-2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+5.0*u(m,i,j,k));
		}

		for (m = 0; m < 5; m++) rtmp[m] *= dt;
	}

	for (m = 0; m < 5; m++) rhs(m,i,j,k) = rtmp[m];
}

//---------------------------------------------------------------------
// adi: txinvr
//---------------------------------------------------------------------
__global__ static void txinvr_kernel (const double *rho_i, const double *us, const double *vs, const double *ws, const double *speed, const double *qs, double *rhs, const int nx, const int ny, const int nz) {
	int i, j, k;

	k = blockIdx.y+1;
	j = blockIdx.x+1;
	i = threadIdx.x+1;

	double ru1 = rho_i(i,j,k);
	double uu = us(i,j,k);
	double vv = vs(i,j,k);
	double ww = ws(i,j,k);
	double ac = speed(i,j,k);
	double ac2inv = 1.0/( ac*ac );

	double r1 = rhs(0,i,j,k);
	double r2 = rhs(1,i,j,k);
	double r3 = rhs(2,i,j,k);
	double r4 = rhs(3,i,j,k);
	double r5 = rhs(4,i,j,k);

	double t1 = c2*ac2inv*(qs(i,j,k)*r1 - uu*r2  - vv*r3 - ww*r4 + r5);
	double t2 = bt * ru1 * ( uu * r1 - r2 );
	double t3 = ( bt * ru1 * ac ) * t1;

	rhs(0,i,j,k) = r1 - t1;
	rhs(1,i,j,k) = -ru1*(ww*r1-r4);
	rhs(2,i,j,k) = ru1*(vv*r1-r3);
	rhs(3,i,j,k) = -t2+t3;
	rhs(4,i,j,k) = t2+t3;
}

//---------------------------------------------------------------------
// adi: x_solve
//---------------------------------------------------------------------
#define lhs(m,i,j,k) lhs[(j-1)+(ny-2)*((k-1)+(nz-2)*((i)+nx*(m-3)))]
#define lhsp(m,i,j,k) lhs[(j-1)+(ny-2)*((k-1)+(nz-2)*((i)+nx*(m+4)))]
#define lhsm(m,i,j,k) lhs[(j-1)+(ny-2)*((k-1)+(nz-2)*((i)+nx*(m-3+2)))]
#define rtmp(m,i,j,k) rhstmp[(j)+ny*((k)+nz*((i)+nx*(m)))]
__global__ static void x_solve_kernel (const double *rho_i, const double *us, const double *speed, double *rhs, double *lhs, double *rhstmp, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rhon[3], cv[3], _lhs[3][5], _lhsp[3][5], _rhs[3][5], fac1;
  double zero;

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	j = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || j >= ny-1) return;

	//---------------------------------------------------------------------
	// Computes the left hand side for the three x-factors  
	//---------------------------------------------------------------------
	//---------------------------------------------------------------------
	//     zap the whole left hand side for starters
	//---------------------------------------------------------------------
	_lhs[0][0] = (double)0.0;
	_lhs[0][1] = (double)0.0;
	_lhs[0][2] = (double)1.0;
	_lhs[0][3] = (double)0.0;
	_lhs[0][4] = (double)0.0;
  lhsp(0,0,j,k) = (double)0.0;
  lhsp(1,0,j,k) = (double)0.0;
  lhsp(2,0,j,k) = (double)1.0;
  lhsp(3,0,j,k) = (double)0.0;
  lhsp(4,0,j,k) = (double)0.0;
  zero = (double)0.0;

	//---------------------------------------------------------------------
	// first fill the lhs for the u-eigenvalue                          
	//--------------------------------------------------------------------
	for (i = 0; i < 3; i++) {
		fac1 = c3c4*rho_i(i,j,k);
		rhon[i] = max(max(max(dx2+con43*fac1, dx5+c1c5*fac1), dxmax+fac1), zero+dx1);
    //if (dx2+con43*fac1>dx5+c1c5*fac1)
    //  rhon[i] = dx2+con43*fac1;
    //else
    //  rhon[i] = dx5+c1c5*fac1;
    //if (rhon[i]<dxmax+fac1)
    //  rhon[i] = dxmax+fac1;
    //if (rhon[i]<zero+dx1)
    //  rhon[i] = zero+dx1;


		cv[i] = us(i,j,k);
	}
	_lhs[1][0] = (double)0.0;
	_lhs[1][1] = - dttx2 * cv[0] - dttx1 * rhon[0];
	_lhs[1][2] = 1.0 + c2dttx1 * rhon[1];
	_lhs[1][3] = dttx2 * cv[2] - dttx1 * rhon[2];
	_lhs[1][4] = (double)0.0;
	_lhs[1][2] += comz5;
	_lhs[1][3] -= comz4;
	_lhs[1][4] += comz1;
	for (m = 0; m < 5; m++) lhsp(m,1,j,k) = _lhs[1][m];
	rhon[0] = rhon[1]; rhon[1] = rhon[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	for (m = 0; m < 3; m++) {
		_rhs[0][m] = rhs(m,0,j,k);
		_rhs[1][m] = rhs(m,1,j,k);
	}

	//---------------------------------------------------------------------
	//      perform the Thomas algorithm; first, FORWARD ELIMINATION     
	//---------------------------------------------------------------------
	for (i = 0; i < nx-2; i++) {
		//---------------------------------------------------------------------
		// first fill the lhs for the u-eigenvalue                          
		//---------------------------------------------------------------------
		if (i+2 == nx-1) {
			_lhs[2][0] = (double)0.0;
			_lhs[2][1] = (double)0.0;
			_lhs[2][2] = (double)1.0;
			_lhs[2][3] = (double)0.0;
			_lhs[2][4] = (double)0.0;
      lhsp(0,i+2,j,k) = (double)0.0;
      lhsp(1,i+2,j,k) = (double)0.0;
      lhsp(2,i+2,j,k) = (double)1.0;
      lhsp(3,i+2,j,k) = (double)0.0;
      lhsp(4,i+2,j,k) = (double)0.0;

		} else {
			fac1 = c3c4*rho_i(i+3,j,k);
			rhon[2] = max(max(max(dx2+con43*fac1, dx5+c1c5*fac1), dxmax+fac1), zero+dx1);
      //if (dx2+con43*fac1>dx5+c1c5*fac1)
      //  rhon[2] = dx2+con43*fac1;
      //else
      //  rhon[2] = dx5+c1c5*fac1;
      //if (rhon[2]<dxmax+fac1)
      //  rhon[2] = dxmax+fac1;
      //if (rhon[2]<zero+dx1)
      //  rhon[2] = zero+dx1;

			cv[2] = us(i+3,j,k);
			_lhs[2][0] = (double)0.0;
			_lhs[2][1] = - dttx2 * cv[0] - dttx1 * rhon[0];
			_lhs[2][2] = 1.0 + c2dttx1 * rhon[1];
			_lhs[2][3] = dttx2 * cv[2] - dttx1 * rhon[2];
			_lhs[2][4] = (double)0.0;
			//---------------------------------------------------------------------
			//      add fourth order dissipation                                  
			//---------------------------------------------------------------------
			if (i+2 == 2) {
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (i+2 >= 3 && i+2 < nx-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (i+2 == nx-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
			} else if (i+2 == nx-2) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz5;
			}

			//---------------------------------------------------------------------
			//      store computed lhs for later reuse
			//---------------------------------------------------------------------
			for (m = 0; m < 5; m++) lhsp(m,i+2,j,k) = _lhs[2][m];
			rhon[0] = rhon[1]; rhon[1] = rhon[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		//---------------------------------------------------------------------
		//      load rhs values for current iteration
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[2][m] = rhs(m,i+2,j,k);

		//---------------------------------------------------------------------
		//      perform current iteration
		//---------------------------------------------------------------------
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		lhs(3,i,j,k) = _lhs[0][3];
		lhs(4,i,j,k) = _lhs[0][4];
		for (m = 0; m < 5; m++) {
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
		for (m = 0; m < 3; m++) {
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
	}

	//---------------------------------------------------------------------
	//      The last two rows in this zone are a bit different, 
	//      since they do not have two more rows available for the
	//      elimination of off-diagonal entries
	//---------------------------------------------------------------------
	i = nx-2;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//            scale the last row immediately 
	//---------------------------------------------------------------------
	fac1 = 1.0/_lhs[1][2];
	for (m = 0; m < 3; m++) _rhs[1][m] *= fac1;
	lhs(3,nx-2,j,k) = _lhs[0][3];
	lhs(4,nx-2,j,k) = _lhs[0][4];

	//---------------------------------------------------------------------
	//      subsequently, fill the other factors (u+c), (u-c) 
	//---------------------------------------------------------------------
	for (i = 0; i < 3; i++) cv[i] = speed(i,j,k);
	for (m = 0; m < 5; m++) {
		_lhsp[0][m] = _lhs[0][m] = lhsp(m,0,j,k);
		_lhsp[1][m] = _lhs[1][m] = lhsp(m,1,j,k);
	}
	_lhsp[1][1] -= dttx2 * cv[0];
	_lhsp[1][3] += dttx2 * cv[2];
	_lhs[1][1] += dttx2 * cv[0];
	_lhs[1][3] -= dttx2 * cv[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	_rhs[0][3] = rhs(3,0,j,k);
	_rhs[0][4] = rhs(4,0,j,k);
	_rhs[1][3] = rhs(3,1,j,k);
	_rhs[1][4] = rhs(4,1,j,k);
	//---------------------------------------------------------------------
	//      do the u+c and the u-c factors               
	//---------------------------------------------------------------------
	for (i = 0; i < nx-2; i++) {
		//---------------------------------------------------------------------
		//      first, fill the other factors (u+c), (u-c) 
		//---------------------------------------------------------------------
		for (m = 0; m < 5; m++) {
			_lhsp[2][m] = _lhs[2][m] = lhsp(m,i+2,j,k);
		}
		_rhs[2][3] = rhs(3,i+2,j,k);
		_rhs[2][4] = rhs(4,i+2,j,k);

		if (i+2 < nx-1) {
			cv[2] = speed(i+3,j,k);
			_lhsp[2][1] -= dttx2 * cv[0];
			_lhsp[2][3] += dttx2 * cv[2];
			_lhs[2][1] += dttx2 * cv[0];
			_lhs[2][3] -= dttx2 * cv[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		m = 3;
		fac1 = 1.0/_lhsp[0][2];
		_lhsp[0][3] *= fac1;
		_lhsp[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhsp[1][2] -= _lhsp[1][1]*_lhsp[0][3];
		_lhsp[1][3] -= _lhsp[1][1]*_lhsp[0][4];
		_rhs[1][m] -= _lhsp[1][1]*_rhs[0][m];
		_lhsp[2][1] -= _lhsp[2][0]*_lhsp[0][3];
		_lhsp[2][2] -= _lhsp[2][0]*_lhsp[0][4];
		_rhs[2][m] -= _lhsp[2][0]*_rhs[0][m];

		m = 4;
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1]*_lhs[0][3];
		_lhs[1][3] -= _lhs[1][1]*_lhs[0][4];
		_rhs[1][m] -= _lhs[1][1]*_rhs[0][m];
		_lhs[2][1] -= _lhs[2][0]*_lhs[0][3];
		_lhs[2][2] -= _lhs[2][0]*_lhs[0][4];
		_rhs[2][m] -= _lhs[2][0]*_rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		for (m = 3; m < 5; m++) {
			lhsp(m,i,j,k) = _lhsp[0][m];
			lhsm(m,i,j,k) = _lhs[0][m];
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
		for (m = 0; m < 5; m++) {
			_lhsp[0][m] = _lhsp[1][m];
			_lhsp[1][m] = _lhsp[2][m];
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//         And again the last two rows separately
	//---------------------------------------------------------------------
	i = nx-2;
	m = 3;
	fac1 = 1.0/_lhsp[0][2];
	_lhsp[0][3] *= fac1;
	_lhsp[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhsp[1][2] -= _lhsp[1][1]*_lhsp[0][3];
	_lhsp[1][3] -= _lhsp[1][1]*_lhsp[0][4];
	_rhs[1][m] -= _lhsp[1][1]*_rhs[0][m];

	m = 4;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1]*_lhs[0][3];
	_lhs[1][3] -= _lhs[1][1]*_lhs[0][4];
	_rhs[1][m] -= _lhs[1][1]*_rhs[0][m];

	//---------------------------------------------------------------------
	//               Scale the last row immediately
	//---------------------------------------------------------------------
	_rhs[1][3] /= _lhsp[1][2];
	_rhs[1][4] /= _lhs[1][2];

	//---------------------------------------------------------------------
	//                         BACKSUBSTITUTION 
	//---------------------------------------------------------------------
	for (m = 0; m < 3; m++) _rhs[0][m] -= lhs(3,nx-2,j,k)*_rhs[1][m];
	_rhs[0][3] -= _lhsp[0][3]*_rhs[1][3];
	_rhs[0][4] -= _lhs[0][3]*_rhs[1][4];
	for (m = 0; m < 5; m++) {
		_rhs[2][m] = _rhs[1][m];
		_rhs[1][m] = _rhs[0][m];
	}

	for (i = nx-3; i >= 0; i--) {
		//---------------------------------------------------------------------
		//      The first three factors
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[0][m] = rtmp(m,i,j,k) - lhs(3,i,j,k)*_rhs[1][m] - lhs(4,i,j,k)*_rhs[2][m];
		//---------------------------------------------------------------------
		//      And the remaining two
		//---------------------------------------------------------------------
		_rhs[0][3] = rtmp(3,i,j,k) - lhsp(3,i,j,k)*_rhs[1][3] - lhsp(4,i,j,k)*_rhs[2][3];
		_rhs[0][4] = rtmp(4,i,j,k) - lhsm(3,i,j,k)*_rhs[1][4] - lhsm(4,i,j,k)*_rhs[2][4];

		if (i+2 < nx-1) {
			//---------------------------------------------------------------------
			//      Do the block-diagonal inversion          
			//---------------------------------------------------------------------
				double r1 = _rhs[2][0];
				double r2 = _rhs[2][1];
				double r3 = _rhs[2][2];
				double r4 = _rhs[2][3];
				double r5 = _rhs[2][4];
				double t1 = bt * r3;
				double t2 = 0.5 * (r4+r5);

				_rhs[2][0] = -r2;
				_rhs[2][1] =  r1;
				_rhs[2][2] = bt * ( r4 - r5 );
				_rhs[2][3] = -t1 + t2;
				_rhs[2][4] =  t1 + t2;
		}

		for (m = 0; m < 5; m++) {
			rhs(m,i+2,j,k) = _rhs[2][m];
			_rhs[2][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[0][m];
		}
	}

	//---------------------------------------------------------------------
	//      Do the block-diagonal inversion          
	//---------------------------------------------------------------------
	double t1 = bt * _rhs[2][2];
	double t2 = 0.5 * (_rhs[2][3]+_rhs[2][4]);
	rhs(0,1,j,k) = -_rhs[2][1];
	rhs(1,1,j,k) =  _rhs[2][0];
	rhs(2,1,j,k) = bt * ( _rhs[2][3] - _rhs[2][4] );
	rhs(3,1,j,k) = -t1 + t2;
	rhs(4,1,j,k) =  t1 + t2;

	for (m = 0; m < 5; m++) rhs(m,0,j,k) = _rhs[1][m];
}
#undef lhs
#undef lhsp
#undef lhsm
#undef rtmp

//---------------------------------------------------------------------
// adi: y_solve
//---------------------------------------------------------------------
#define lhs(m,i,j,k) lhs[(i-1)+(nx-2)*((k-1)+(nz-2)*((j)+ny*(m-3)))]
#define lhsp(m,i,j,k) lhs[(i-1)+(nx-2)*((k-1)+(nz-2)*((j)+ny*(m+4)))]
#define lhsm(m,i,j,k) lhs[(i-1)+(nx-2)*((k-1)+(nz-2)*((j)+ny*(m-3+2)))]
#define rtmp(m,i,j,k) rhstmp[(i)+nx*((k)+nz*((j)+ny*(m)))]
__global__ static void y_solve_kernel (const double *rho_i, const double *vs, const double *speed, double *rhs, double *lhs, double *rhstmp, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rhoq[3], cv[3], _lhs[3][5], _lhsp[3][5], _rhs[3][5], fac1;
  double zero;

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || i >= nx-1) return;

	//---------------------------------------------------------------------
	// Computes the left hand side for the three y-factors   
	//---------------------------------------------------------------------
	//---------------------------------------------------------------------
	//     zap the whole left hand side for starters
	//---------------------------------------------------------------------
	_lhs[0][0] = (double)0.0;
	_lhs[0][1] = (double)0.0;
	_lhs[0][2] = (double)1.0;
	_lhs[0][3] = (double)0.0;
	_lhs[0][4] = (double)0.0;
  lhsp(0,i,0,k) = (double)0.0;
  lhsp(1,i,0,k) = (double)0.0;
  lhsp(2,i,0,k) = (double)1.0;
  lhsp(3,i,0,k) = (double)0.0;
  lhsp(4,i,0,k) = (double)0.0;
  zero = (double)0.0;


	//---------------------------------------------------------------------
	//      first fill the lhs for the u-eigenvalue         
	//---------------------------------------------------------------------
	for (j = 0; j < 3; j++) {
		fac1 = c3c4*rho_i(i,j,k);
		rhoq[j] = max(max(max(dy3+con43*fac1, dy5+c1c5*fac1), dymax+fac1), zero+dy1);
    //if (dy3+con43*fac1>dy5+c1c5*fac1)
    //  rhoq[j] = dy3+con43*fac1;
    //else
    //  rhoq[j] = dy5+c1c5*fac1;
    //if (rhoq[j]<dymax+fac1)
    //  rhoq[j] = dymax+fac1;
    //if (rhoq[j]<zero+dy1)
    //  rhoq[j] = zero+dy1;
		cv[j] = vs(i,j,k);
	}
	_lhs[1][0] =  (double)0.0;
	_lhs[1][1] = -dtty2*cv[0]-dtty1 * rhoq[0];
	_lhs[1][2] =  1.0 + c2dtty1 * rhoq[1];
	_lhs[1][3] =  dtty2*cv[2]-dtty1 * rhoq[2];
	_lhs[1][4] =  (double)0.0;
	_lhs[1][2] += comz5;
	_lhs[1][3] -= comz4;
	_lhs[1][4] += comz1;
	for (m = 0; m < 5; m++) lhsp(m,i,1,k) = _lhs[1][m];
	rhoq[0] = rhoq[1]; rhoq[1] = rhoq[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	for (m = 0; m < 3; m++) {
		_rhs[0][m] = rhs(m,i,0,k);
		_rhs[1][m] = rhs(m,i,1,k);
	}

	//---------------------------------------------------------------------
	//                          FORWARD ELIMINATION  
	//---------------------------------------------------------------------
	for (j = 0; j < ny-2; j++) {
		//---------------------------------------------------------------------
		// first fill the lhs for the u-eigenvalue                          
		//---------------------------------------------------------------------
		if (j+2 == ny-1) {
			_lhs[2][0] = (double)0.0;
			_lhs[2][1] = (double)0.0;
			_lhs[2][2] = (double)1.0;
			_lhs[2][3] = (double)0.0;
			_lhs[2][4] = (double)0.0;
      lhsp(0,i,j+2,k) = (double)0.0;
      lhsp(1,i,j+2,k) = (double)0.0;
      lhsp(2,i,j+2,k) = (double)1.0;
      lhsp(3,i,j+2,k) = (double)0.0;
      lhsp(4,i,j+2,k) = (double)0.0;
		} else {
			fac1 = c3c4*rho_i(i,j+3,k);
			rhoq[2] = max(max(max(dy3+con43*fac1, dy5+c1c5*fac1), dymax+fac1), zero+dy1);
      //if (dy3+con43*fac1>dy5+c1c5*fac1)
      //  rhoq[2] = dy3+con43*fac1;
      //else
      //  rhoq[2] = dy5+c1c5*fac1;
      //if (rhoq[2]<dymax+fac1)
      //  rhoq[2] = dymax+fac1;
      //if (rhoq[2]<zero+dy1)
      //  rhoq[2] = zero+dy1;
			cv[2] = vs(i,j+3,k);
			_lhs[2][0] =  (double)0.0;
			_lhs[2][1] = -dtty2*cv[0]-dtty1 * rhoq[0];
			_lhs[2][2] =  1.0 + c2dtty1 * rhoq[1];
			_lhs[2][3] =  dtty2*cv[2]-dtty1 * rhoq[2];
			_lhs[2][4] =  (double)0.0;
			//---------------------------------------------------------------------
			//      add fourth order dissipation                             
			//---------------------------------------------------------------------
			if (j+2 == 2) {
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (j+2 >= 3 && j+2 < ny-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (j+2 == ny-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
			} else if (j+2 == ny-2) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz5;
			}

			//---------------------------------------------------------------------
			//      store computed lhs for later reuse
			//---------------------------------------------------------------------
			for (m = 0; m < 5; m++) lhsp(m,i,j+2,k) = _lhs[2][m];
			rhoq[0] = rhoq[1]; rhoq[1] = rhoq[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		//---------------------------------------------------------------------
		//      load rhs values for current iteration
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[2][m] = rhs(m,i,j+2,k);

		//---------------------------------------------------------------------
		//      perform current iteration
		//---------------------------------------------------------------------
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		lhs(3,i,j,k) = _lhs[0][3];
		lhs(4,i,j,k) = _lhs[0][4];
		for (m = 0; m < 5; m++) {
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
		for (m = 0; m < 3; m++) {
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//      The last two rows in this zone are a bit different, 
	//      since they do not have two more rows available for the
	//      elimination of off-diagonal entries
	//---------------------------------------------------------------------
	j = ny-2;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//            scale the last row immediately 
	//---------------------------------------------------------------------
	fac1 = 1.0/_lhs[1][2];
	for (m = 0; m < 3; m++) _rhs[1][m] *= fac1;
	lhs(3,i,ny-2,k) = _lhs[0][3];
	lhs(4,i,ny-2,k) = _lhs[0][4];

	//---------------------------------------------------------------------
	//      do the u+c and the u-c factors                 
	//---------------------------------------------------------------------
	for (j = 0; j < 3; j++) cv[j] = speed(i,j,k);
	for (m = 0; m < 5; m++) {
		_lhsp[0][m] = _lhs[0][m] = lhsp(m,i,0,k);
		_lhsp[1][m] = _lhs[1][m] = lhsp(m,i,1,k);
	}
	_lhsp[1][1] -= dtty2*cv[0];
	_lhsp[1][3] += dtty2*cv[2];
	_lhs[1][1] += dtty2*cv[0];
	_lhs[1][3] -= dtty2*cv[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	_rhs[0][3] = rhs(3,i,0,k);
	_rhs[0][4] = rhs(4,i,0,k);
	_rhs[1][3] = rhs(3,i,1,k);
	_rhs[1][4] = rhs(4,i,1,k);
	for (j = 0; j < ny-2; j++) {
		for (m = 0; m < 5; m++) {
			_lhsp[2][m] = _lhs[2][m] = lhsp(m,i,j+2,k);
		}
		_rhs[2][3] = rhs(3,i,j+2,k);
		_rhs[2][4] = rhs(4,i,j+2,k);
		if (j+2 < ny-1) {
			cv[2] = speed(i,j+3,k);
			_lhsp[2][1] -= dtty2*cv[0];
			_lhsp[2][3] += dtty2*cv[2];
			_lhs[2][1] += dtty2*cv[0];
			_lhs[2][3] -= dtty2*cv[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		fac1 = 1.0/_lhsp[0][2];
		m = 3;
		_lhsp[0][3] *= fac1;
		_lhsp[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhsp[1][2] -= _lhsp[1][1] * _lhsp[0][3];
		_lhsp[1][3] -= _lhsp[1][1] * _lhsp[0][4];
		_rhs[1][m] -= _lhsp[1][1] * _rhs[0][m];
		_lhsp[2][1] -= _lhsp[2][0] * _lhsp[0][3];
		_lhsp[2][2] -= _lhsp[2][0] * _lhsp[0][4];
		_rhs[2][m] -= _lhsp[2][0] * _rhs[0][m];

		m = 4;
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		_rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		_rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		for (m = 3; m < 5; m++) {
			lhsp(m,i,j,k) = _lhsp[0][m];
			lhsm(m,i,j,k) = _lhs[0][m];
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
		for (m = 0; m < 5; m++) {
			_lhsp[0][m] = _lhsp[1][m];
			_lhsp[1][m] = _lhsp[2][m];
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//         And again the last two rows separately
	//---------------------------------------------------------------------
	j = ny-2;
	m = 3;
	fac1 = 1.0/_lhsp[0][2];
	_lhsp[0][3] *= fac1;
	_lhsp[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhsp[1][2] -= _lhsp[1][1] * _lhsp[0][3];
	_lhsp[1][3] -= _lhsp[1][1] * _lhsp[0][4];
	_rhs[1][m] -= _lhsp[1][1] * _rhs[0][m];

	m = 4;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	_rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//               Scale the last row immediately 
	//---------------------------------------------------------------------
	_rhs[1][3] /= _lhsp[1][2];
	_rhs[1][4] /= _lhs[1][2];

	//---------------------------------------------------------------------
	//                         BACKSUBSTITUTION 
	//---------------------------------------------------------------------
	for (m = 0; m < 3; m++) _rhs[0][m] -= lhs(3,i,ny-2,k) * _rhs[1][m];
	_rhs[0][3] -= _lhsp[0][3] * _rhs[1][3];
	_rhs[0][4] -= _lhs[0][3] * _rhs[1][4];
	for (m = 0; m < 5; m++) {
		_rhs[2][m] = _rhs[1][m];
		_rhs[1][m] = _rhs[0][m];
	}
	for (j = ny-3; j >= 0; j--) {
		//---------------------------------------------------------------------
		//      The first three factors
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[0][m] = rtmp(m,i,j,k) - lhs(3,i,j,k)*_rhs[1][m] - lhs(4,i,j,k)*_rhs[2][m];
		//---------------------------------------------------------------------
		//      And the remaining two
		//---------------------------------------------------------------------
		_rhs[0][3] = rtmp(3,i,j,k) - lhsp(3,i,j,k)*_rhs[1][3] - lhsp(4,i,j,k)*_rhs[2][3];
		_rhs[0][4] = rtmp(4,i,j,k) - lhsm(3,i,j,k)*_rhs[1][4] - lhsm(4,i,j,k)*_rhs[2][4];
	
		if (j+2 < ny-1) {
			//---------------------------------------------------------------------
			//   block-diagonal matrix-vector multiplication                       
			//---------------------------------------------------------------------
			double r1 = _rhs[2][0];
			double r2 = _rhs[2][1];
			double r3 = _rhs[2][2];
			double r4 = _rhs[2][3];
			double r5 = _rhs[2][4];

			double t1 = bt * r1;
			double t2 = 0.5 * ( r4 + r5 );

			_rhs[2][0] =  bt * ( r4 - r5 );
			_rhs[2][1] = -r3;
			_rhs[2][2] =  r2;
			_rhs[2][3] = -t1 + t2;
			_rhs[2][4] =  t1 + t2;
		}

		for (m = 0; m < 5; m++) {
			rhs(m,i,j+2,k) = _rhs[2][m];
			_rhs[2][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[0][m];
		}
	}

	//---------------------------------------------------------------------
	//   block-diagonal matrix-vector multiplication                       
	//---------------------------------------------------------------------
	double t1 = bt * _rhs[2][0];
	double t2 = 0.5 * ( _rhs[2][3] + _rhs[2][4] );
	rhs(0,i,1,k) =  bt * ( _rhs[2][3] - _rhs[2][4] );
	rhs(1,i,1,k) = -_rhs[2][2];
	rhs(2,i,1,k) =  _rhs[2][1];
	rhs(3,i,1,k) = -t1 + t2;
	rhs(4,i,1,k) =  t1 + t2;

	for (m = 0; m < 5; m++) rhs(m,i,0,k) = _rhs[1][m];
}
#undef lhs
#undef lhsp
#undef lhsm
#undef rtmp

//---------------------------------------------------------------------
// adi: z_solve
//---------------------------------------------------------------------
#define lhs(m,i,j,k) lhs[(i-1)+(nx-2)*((j-1)+(ny-2)*((k)+nz*(m-3)))]
#define lhsp(m,i,j,k) lhs[(i-1)+(nx-2)*((j-1)+(ny-2)*((k)+nz*(m+4)))]
#define lhsm(m,i,j,k) lhs[(i-1)+(nx-2)*((j-1)+(ny-2)*((k)+nz*(m-3+2)))]
#define rtmp(m,i,j,k) rhstmp[(i)+nx*((j)+ny*((k)+nz*(m)))]
__global__ static void z_solve_kernel (const double *rho_i, const double *us, const double *vs, const double *ws, const double *speed, const double *qs, const double *u, double *rhs, double *lhs, double *rhstmp, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rhos[3], cv[3], _lhs[3][5], _lhsp[3][5], _rhs[3][5], fac1;
  double zero;

	j = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (j >= ny-1 || i >= nx-1) return;

	//---------------------------------------------------------------------
	// Computes the left hand side for the three z-factors   
	//---------------------------------------------------------------------
	//---------------------------------------------------------------------
	//     zap the whole left hand side for starters
	//---------------------------------------------------------------------
	_lhs[0][0] = (double)0.0;
	_lhs[0][1] = (double)0.0;
	_lhs[0][2] = (double)1.0;
	_lhs[0][3] = (double)0.0;
	_lhs[0][4] = (double)0.0;
  lhsp(0,i,j,0) = (double)0.0;
  lhsp(1,i,j,0) = (double)0.0;
  lhsp(2,i,j,0) = (double)1.0;
  lhsp(3,i,j,0) = (double)0.0;
  lhsp(4,i,j,0) = (double)0.0;
  zero = (double)0.0;

	//---------------------------------------------------------------------
	// first fill the lhs for the u-eigenvalue                          
	//---------------------------------------------------------------------
	for (k = 0; k < 3; k++) {
		fac1 = c3c4*rho_i(i,j,k);
		rhos[k] = max(max(max(dz4+con43*fac1, dz5+c1c5*fac1), dzmax+fac1), zero+dz1);
    //if (dz4+con43*fac1>dx5+c1c5*fac1)
    //  rhos[k] = dz4+con43*fac1;
    //else
    //  rhos[k] = dz5+c1c5*fac1;
    //if (rhos[k]<dzmax+fac1)
    //  rhos[k] = dzmax+fac1;
    //if (rhos[k]<zero+dz1)
    //  rhos[k] = zero+dz1;
		cv[k] = ws(i,j,k);
	}
	_lhs[1][0] =  (double)0.0;
	_lhs[1][1] = -dttz2*cv[0] - dttz1*rhos[0];
	_lhs[1][2] =  1.0 + c2dttz1 * rhos[1];
	_lhs[1][3] =  dttz2*cv[2] - dttz1*rhos[2];
	_lhs[1][4]=  (double)0.0;
	_lhs[1][2] += comz5;
	_lhs[1][3] -= comz4;
	_lhs[1][4] += comz1;
	for (m = 0; m < 5; m++) lhsp(m,i,j,1) = _lhs[1][m];
	rhos[0] = rhos[1]; rhos[1] = rhos[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	for (m = 0; m < 3; m++) {
		_rhs[0][m] = rhs(m,i,j,0);
		_rhs[1][m] = rhs(m,i,j,1);
	}

	//---------------------------------------------------------------------
	//                          FORWARD ELIMINATION  
	//---------------------------------------------------------------------
	for (k = 0; k < nz-2; k++) {
		//---------------------------------------------------------------------
		// first fill the lhs for the u-eigenvalue                          
		//---------------------------------------------------------------------
		if (k+2 == nz-1) {
			_lhs[2][0] = (double)0.0;
			_lhs[2][1] = (double)0.0;
			_lhs[2][2] = (double)1.0;
			_lhs[2][3] = (double)0.0;
			_lhs[2][4] = (double)0.0;
      lhsp(0,i,j,k+2) = (double)0.0;
      lhsp(1,i,j,k+2) = (double)0.0;
      lhsp(2,i,j,k+2) = (double)1.0;
      lhsp(3,i,j,k+2) = (double)0.0;
      lhsp(4,i,j,k+2) = (double)0.0;
		} else {
			fac1 = c3c4*rho_i(i,j,k+3);
			rhos[2] = max(max(max(dz4+con43*fac1, dz5+c1c5*fac1), dzmax+fac1), zero+dz1);
      //if (dz4+con43*fac1>dx5+c1c5*fac1)
      //  rhos[2] = dz4+con43*fac1;
      //else
      //  rhos[2] = dz5+c1c5*fac1;
      //if (rhos[2]<dzmax+fac1)
      //  rhos[2] = dzmax+fac1;
      //if (rhos[2]<zero+dz1)
      //  rhos[2] = zero+dz1;
			cv[2] = ws(i,j,k+3);
			_lhs[2][0] =  (double)0.0;
			_lhs[2][1] = -dttz2*cv[0] - dttz1*rhos[0];
			_lhs[2][2] =  1.0 + c2dttz1 * rhos[1];
			_lhs[2][3] =  dttz2*cv[2] - dttz1*rhos[2];
			_lhs[2][4] =  (double)0.0;
			//---------------------------------------------------------------------
			//      add fourth order dissipation                                  
			//---------------------------------------------------------------------
			if (k+2 == 2) {
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (k+2 >= 3 && k+2 < nz-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (k+2 == nz-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
			} else if (k+2 == nz-2) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz5;
			}

			//---------------------------------------------------------------------
			//      store computed lhs for later reuse
			//---------------------------------------------------------------------
			for (m = 0; m < 5; m++) lhsp(m,i,j,k+2) = _lhs[2][m];
			rhos[0] = rhos[1]; rhos[1] = rhos[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		//---------------------------------------------------------------------
		//      load rhs values for current iteration
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[2][m] = rhs(m,i,j,k+2);

		//---------------------------------------------------------------------
		//      perform current iteration
		//---------------------------------------------------------------------
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		lhs(3,i,j,k) = _lhs[0][3];
		lhs(4,i,j,k) = _lhs[0][4];
		for (m = 0; m < 5; m++) {
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
		for (m = 0; m < 3; m++) {
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//      The last two rows in this zone are a bit different, 
	//      since they do not have two more rows available for the
	//      elimination of off-diagonal entries
	//---------------------------------------------------------------------
	k = nz-2;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//               scale the last row immediately
	//---------------------------------------------------------------------
	fac1 = 1.0/_lhs[1][2];
	for (m = 0; m < 3; m++) _rhs[1][m] *= fac1;
	lhs(3,i,j,k) = _lhs[0][3];
	lhs(4,i,j,k) = _lhs[0][4];

	//---------------------------------------------------------------------
	//      subsequently, fill the other factors (u+c), (u-c) 
	//---------------------------------------------------------------------
	for (k = 0; k < 3; k++) cv[k] = speed(i,j,k);
	for (m = 0; m < 5; m++) {
		_lhsp[0][m] = _lhs[0][m] = lhsp(m,i,j,0);
		_lhsp[1][m] = _lhs[1][m] = lhsp(m,i,j,1);
	}
	_lhsp[1][1] -= dttz2*cv[0];
	_lhsp[1][3] += dttz2*cv[2];
	_lhs[1][1] += dttz2*cv[0];
	_lhs[1][3] -= dttz2*cv[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	_rhs[0][3] = rhs(3,i,j,0);
	_rhs[0][4] = rhs(4,i,j,0);
	_rhs[1][3] = rhs(3,i,j,1);
	_rhs[1][4] = rhs(4,i,j,1);
	//---------------------------------------------------------------------
	//      do the u+c and the u-c factors               
	//---------------------------------------------------------------------
	for (k = 0; k < nz-2; k++) {
		//---------------------------------------------------------------------
		//      first, fill the other factors (u+c), (u-c) 
		//---------------------------------------------------------------------
		for (m = 0; m < 5; m++) {
			_lhsp[2][m] = _lhs[2][m] = lhsp(m,i,j,k+2);
		}
		_rhs[2][3] = rhs(3,i,j,k+2);
		_rhs[2][4] = rhs(4,i,j,k+2);
		if (k+2 < nz-1) {
			cv[2] = speed(i,j,k+3);
			_lhsp[2][1] -= dttz2*cv[0];
			_lhsp[2][3] += dttz2*cv[2];
			_lhs[2][1] += dttz2*cv[0];
			_lhs[2][3] -= dttz2*cv[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		m = 3;
		fac1 = 1.0/_lhsp[0][2];
		_lhsp[0][3] *= fac1;
		_lhsp[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhsp[1][2] -= _lhsp[1][1] * _lhsp[0][3];
		_lhsp[1][3] -= _lhsp[1][1] * _lhsp[0][4];
		_rhs[1][m] -= _lhsp[1][1] * _rhs[0][m];
		_lhsp[2][1] -= _lhsp[2][0] * _lhsp[0][3];
		_lhsp[2][2] -= _lhsp[2][0] * _lhsp[0][4];
		_rhs[2][m] -= _lhsp[2][0] * _rhs[0][m];

		m = 4;
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		_rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		_rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		for (m = 3; m < 5; m++) {
			lhsp(m,i,j,k) = _lhsp[0][m];
			lhsm(m,i,j,k) = _lhs[0][m];
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
		for (m = 0; m < 5; m++) {
			_lhsp[0][m] = _lhsp[1][m];
			_lhsp[1][m] = _lhsp[2][m];
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//         And again the last two rows separately
	//---------------------------------------------------------------------
	k = nz-2;
	m = 3;
	fac1 = 1.0/_lhsp[0][2];
	_lhsp[0][3] *= fac1;
	_lhsp[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhsp[1][2] -= _lhsp[1][1] * _lhsp[0][3];
	_lhsp[1][3] -= _lhsp[1][1] * _lhsp[0][4];
	_rhs[1][m] -= _lhsp[1][1] * _rhs[0][m];

	m = 4;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	_rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//               Scale the last row immediately (some of this is overkill
	//               if this is the last cell)
	//---------------------------------------------------------------------
	_rhs[1][3] /= _lhsp[1][2];
	_rhs[1][4] /= _lhs[1][2];
		
	//---------------------------------------------------------------------
	//                         BACKSUBSTITUTION 
	//---------------------------------------------------------------------
	for (m = 0; m < 3; m++) _rhs[0][m] -= lhs(3,i,j,nz-2) * _rhs[1][m];
	_rhs[0][3] -= _lhsp[0][3] * _rhs[1][3];
	_rhs[0][4] -= _lhs[0][3] * _rhs[1][4];
	for (m = 0; m < 5; m++) {
		_rhs[2][m] = _rhs[1][m];
		_rhs[1][m] = _rhs[0][m];
	}
	
	for (k = nz-3; k >= 0; k--) {
		//---------------------------------------------------------------------
		//      The first three factors
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[0][m] = rtmp(m,i,j,k) - lhs(3,i,j,k)*_rhs[1][m] - lhs(4,i,j,k)*_rhs[2][m];
		//---------------------------------------------------------------------
		//      And the remaining two
		//---------------------------------------------------------------------
		_rhs[0][3] = rtmp(3,i,j,k) - lhsp(3,i,j,k)*_rhs[1][3] - lhsp(4,i,j,k)*_rhs[2][3];
		_rhs[0][4] = rtmp(4,i,j,k) - lhsm(3,i,j,k)*_rhs[1][4] - lhsm(4,i,j,k)*_rhs[2][4];

		if (k+2 < nz-1) {
			//---------------------------------------------------------------------
			//   block-diagonal matrix-vector multiplication (tzetar)
			//---------------------------------------------------------------------
			double xvel = us(i,j,k+2);
			double yvel = vs(i,j,k+2);
			double zvel = ws(i,j,k+2);
			double ac = speed(i,j,k+2);
			double uzik1 = u(0,i,j,k+2);
			double t1 = (bt*uzik1)/ac * (_rhs[2][3] + _rhs[2][4]);
			double t2 = _rhs[2][2] + t1;
			double t3 = bt*uzik1 * (_rhs[2][3] - _rhs[2][4]);

			_rhs[2][4] =  uzik1*(-xvel*_rhs[2][1] + yvel*_rhs[2][0]) + qs(i,j,k+2)*t2 + c2iv*(ac*ac)*t1 + zvel*t3;
			_rhs[2][3] =  zvel*t2  + t3;
			_rhs[2][2] =  uzik1*_rhs[2][0] + yvel*t2;
			_rhs[2][1] = -uzik1*_rhs[2][1] + xvel*t2;
			_rhs[2][0] = t2;
		}

		for (m = 0; m < 5; m++) {
			rhs(m,i,j,k+2) = _rhs[2][m];
			_rhs[2][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[0][m];
		}
	}

	//---------------------------------------------------------------------
	//   block-diagonal matrix-vector multiplication (tzetar)
	//---------------------------------------------------------------------
	double xvel = us(i,j,1);
	double yvel = vs(i,j,1);
	double zvel = ws(i,j,1);
	double ac = speed(i,j,1);
	double uzik1 = u(0,i,j,1);
	double t1 = (bt*uzik1)/ac * (_rhs[2][3] + _rhs[2][4]);
	double t2 = _rhs[2][2] + t1;
	double t3 = bt*uzik1 * (_rhs[2][3] - _rhs[2][4]);

	rhs(4,i,j,1) =  uzik1*(-xvel*_rhs[2][1] + yvel*_rhs[2][0]) + qs(i,j,1)*t2 + c2iv*(ac*ac)*t1 + zvel*t3;
	rhs(3,i,j,1) =  zvel*t2  + t3;
	rhs(2,i,j,1) =  uzik1*_rhs[2][0] + yvel*t2;
	rhs(1,i,j,1) = -uzik1*_rhs[2][1] + xvel*t2;
	rhs(0,i,j,1) = t2;

	for (m = 0; m < 5; m++) rhs(m,i,j,0) = _rhs[1][m];
}
#undef lhs
#undef lhsp
#undef lhsm
#undef rtmp
//---------------------------------------------------------------------
// 	addition of update to the vector u
//---------------------------------------------------------------------
__global__ static void add_kernel (double *u, const double *rhs, const int nx, const int ny, const int nz) {
	int i, j, k, m;

	k = blockIdx.y+1;
	j = blockIdx.x+1;
	i = threadIdx.x+1;
	m = threadIdx.y;

	u(m,i,j,k) += rhs(m,i,j,k);
}

//---------------------------------------------------------------------
// adi
//---------------------------------------------------------------------
void adi(bool singlestep, int nx, int ny, int nz, int niter, double* rho_i, double* us, double* vs, double* ws, 
         double* speed, double* qs, double* square, double* rhs, double* lhs, double* forcing, double* u, double* rhstmp) {

	HANDLE_ERROR(hipDeviceSynchronize());

	int itmax = singlestep ? 1 : niter;
  int xblock, xgrid, yblock, ygrid, zblock, zgrid;
	for (int step = 1; step <= itmax; step++) {
		if (step % 20 == 0 || step == 1 && !singlestep)
			printf(" Time step %4d\n", step);

		//compute_rhs();
	  dim3 grid1(ny,nz);
	  compute_rhs_kernel_1<<<grid1,nx>>>(rho_i, us, vs, ws, speed, qs, square, u, nx, ny, nz);

	  compute_rhs_kernel_2<<<grid1,nx>>>(rho_i, us, vs, ws, qs, square, rhs, forcing, u, nx, ny, nz);
		//txinvr();
	  dim3 grid2(ny-2,nz-2);
	  txinvr_kernel<<<grid2,nx-2>>> (rho_i, us, vs, ws, speed, qs, rhs, nx, ny, nz);

		//x_solve();
	  yblock = min(SOLVE_BLOCK,ny);
	  ygrid = (ny+yblock-1)/yblock;
	  zblock = min(SOLVE_BLOCK/yblock,nz);
	  zgrid = (nz+zblock-1)/zblock;
	  dim3 grid3(zgrid,ygrid), block3(zblock,yblock);
	  x_solve_kernel<<<grid3,block3>>>(rho_i, us, speed, rhs, lhs, rhstmp, nx, ny, nz);

		//y_solve();
	  xblock = min(SOLVE_BLOCK,nx);
	  xgrid = (nx+xblock-1)/xblock;
	  zblock = min(SOLVE_BLOCK/xblock,nz);
	  zgrid = (nz+zblock-1)/zblock;
	  dim3 grid4(zgrid,xgrid), block4(zblock,xblock);
	  y_solve_kernel<<<grid4,block4>>>(rho_i, vs, speed, rhs, lhs, rhstmp, nx, ny, nz);

		//z_solve();
	  xblock = min(SOLVE_BLOCK,nx);
	  xgrid = (nx+xblock-1)/xblock;
	  yblock = min(SOLVE_BLOCK/xblock,ny);
	  ygrid = (ny+yblock-1)/yblock;
	  dim3 grid5(ygrid,xgrid), block5(yblock,xblock);
	  z_solve_kernel<<<grid5,block5>>>(rho_i, us, vs, ws, speed, qs, u, rhs, lhs, rhstmp, nx, ny, nz);
		//add();
	  dim3 grid6(ny-2,nz-2);
	  dim3 block6(nx-2,5);
	  add_kernel<<<grid6,block6>>>(u, rhs, nx, ny, nz);
	}

	HANDLE_ERROR(hipDeviceSynchronize());
}

//---------------------------------------------------------------------
//     this function computes the norm of the difference between the
//     computed solution and the exact solution
//---------------------------------------------------------------------
//__global__ static void error_norm_kernel (double *rms, const double *u, const int nx, const int ny, const int nz) {
//	int i, j, k, m;
//	double xi, eta, zeta, u_exact[5], rms_loc[5];
//
//	j = blockIdx.x*blockDim.x+threadIdx.x;
//	i = blockIdx.y*blockDim.y+threadIdx.y;
//	if (j >= ny || i >= nx) return;
//
//	for (m = 0; m < 5; m++) rms_loc[m] = 0.0;
//
//	xi = (double)i * dnxm1;
//	eta = (double)j * dnym1;
//
//	for (k = 0; k < nz; k++) {
//		zeta = (double)k * dnzm1;
//		exact_solution_kernel (xi, eta, zeta, u_exact);
//		for (m = 0; m < 5; m++) {
//			double add = u(m,i,j,k) - u_exact[m];
//			rms_loc[m] += add*add;
//		}
//	}
//
//	for (m = 0; m < 5; m++) rms[i+nx*(j+ny*m)] = rms_loc[m];
//}
//
//__global__ static void reduce_norm_kernel (double *rms, const int nx, const int ny, const int nz) {
//	int i, m, maxpos, dist;
//	__shared__ double buffer[NORM_BLOCK][5];
//
//	i = threadIdx.x;
//	for (m = 0; m < 5; m++) buffer[i][m] = 0.0;
//
//	while (i < nx*ny) {
//		for (m = 0; m < 5; m++) buffer[threadIdx.x][m] += rms[i+nx*ny*m];
//		i += blockDim.x;
//	}
//
//	maxpos = blockDim.x;
//	dist = (maxpos+1)/2;
//	i = threadIdx.x;
//	__syncthreads();
//	while (maxpos > 1) {
//		if (i < dist && i+dist < maxpos)
//			for (m = 0; m < 5; m++) buffer[i][m] += buffer[i+dist][m];
//		maxpos = dist;
//		dist = (dist+1)/2;
//		__syncthreads();
//	}
//	
//	m = threadIdx.x;
//	if (m < 5) rms[m] = sqrt(buffer[0][m]/((double)(nz-2)*(double)(ny-2)*(double)(nx-2)));
//}
//
//void error_norm (double* rmsbuf, double* u, double* xce, int nx, int ny, int nz) {
//	int xblock = min(64,nx);
//	int xgrid = (nx+xblock-1)/xblock;
//	int yblock = min(64/xblock,ny);
//	int ygrid = (ny+yblock-1)/yblock;
//	dim3 grid(ygrid,xgrid), block(yblock,xblock);
//
//	error_norm_kernel<<<grid,block>>>(rmsbuf, u, nx, ny, nz);
//	reduce_norm_kernel<<<1,NORM_BLOCK>>>(rmsbuf, nx, ny, nz);
//	HANDLE_ERROR(cudaMemcpy(xce, rmsbuf, 5*sizeof(double), cudaMemcpyDeviceToHost));
//}
//
//__global__ static void rhs_norm_kernel (double *rms, const double *rhs, const int nx, const int ny, const int nz) {
//	int i, j, k, m;
//	double rms_loc[5];
//
//	j = blockIdx.x*blockDim.x+threadIdx.x;
//	i = blockIdx.y*blockDim.y+threadIdx.y;
//	if (j >= ny || i >= nx) return;
//
//	for (m = 0; m < 5; m++) rms_loc[m] = 0.0;
//	if (i >= 1 && i < nx-1 && j >= 1 && j < ny-1) {
//		for (k = 1; k < nz-1; k++) {
//			for (int m = 0; m < 5; m++) {
//				double add = rhs(m,i,j,k);
//				rms_loc[m] += add*add;
//			}
//		}
//	}
//
//	for (m = 0; m < 5; m++) rms[i+nx*(j+ny*m)] = rms_loc[m];
//
//}
//void rhs_norm (double* rmsbuf, double* rhs, double* xcr, int nx, int ny, int nz) {
//	int xblock = min(64,nx);
//	int xgrid = (nx+xblock-1)/xblock;
//	int yblock = min(64/xblock,ny);
//	int ygrid = (ny+yblock-1)/yblock;
//	dim3 grid(ygrid,xgrid), block(yblock,xblock);
//
//	rhs_norm_kernel<<<grid,block>>>(rmsbuf, rhs, nx, ny, nz);
//	reduce_norm_kernel<<<1,NORM_BLOCK>>>(rmsbuf, nx, ny, nz);
//	HANDLE_ERROR(cudaMemcpy(xcr, rmsbuf, 5*sizeof(double), cudaMemcpyDeviceToHost));
//}
//////---------------------------------------------------------------------
//////  verification routine                         
//////---------------------------------------------------------------------
//	bool verified = true;
//	*verifyclass = 'U';
//
//	//---------------------------------------------------------------------
//	//   tolerance level
//	//---------------------------------------------------------------------
//	double epsilon = 1.0e-08;
//
//	//---------------------------------------------------------------------
//	//   compute the error norm and the residual norm, and exit if not printing
//	//---------------------------------------------------------------------
//	error_norm(rmsbuf, u, xce, nx, ny, nz);
//
//	//compute_rhs();
//	dim3 grid1(ny,nz);
//	compute_rhs_kernel_1<<<grid1,nx>>>(rho_i, us, vs, ws, speed, qs, square, u, nx, ny, nz);
//	compute_rhs_kernel_2<<<grid1,nx>>>(rho_i, us, vs, ws, qs, square, rhs, forcing, u, nx, ny, nz);
//
//	rhs_norm(rmsbuf, rhs, xcr, nx, ny, nz);
//
//
//	double xcrref[5], xceref[5], dtref;
//	for (int m = 0; m < 5; m++) xcrref[m] = xceref[m] = 1.0;
//	dtref = 1.0;
//
//	if (nx == 12 && ny == 12 && nz == 12 && niter == 100) {
//		//---------------------------------------------------------------------
//		//    reference data for 12X12X12 grids after 100 time steps, with DT = 1.50d-02
//		//---------------------------------------------------------------------
//		*verifyclass = 'S';
//		dtref = 1.5e-2;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of residual.
//		//---------------------------------------------------------------------
//		xcrref[0] = 2.7470315451339479e-02;
//		xcrref[1] = 1.0360746705285417e-02;
//		xcrref[2] = 1.6235745065095532e-02;
//		xcrref[3] = 1.5840557224455615e-02;
//		xcrref[4] = 3.4849040609362460e-02;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of solution error.
//		//---------------------------------------------------------------------
//		xceref[0] = 2.7289258557377227e-05;
//		xceref[1] = 1.0364446640837285e-05;
//		xceref[2] = 1.6154798287166471e-05;
//		xceref[3] = 1.5750704994480102e-05;
//		xceref[4] = 3.4177666183390531e-05;
//	} else if (nx == 36 && ny == 36 && nz == 36 && niter == 400) {
//		//---------------------------------------------------------------------
//		//    reference data for 36X36X36 grids after 400 time steps, with DT = 1.5d-03
//		//---------------------------------------------------------------------
//		*verifyclass = 'W';
//		dtref = 1.5e-3;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of residual.
//		//---------------------------------------------------------------------
//		xcrref[0] = 0.1893253733584e-02;
//		xcrref[1] = 0.1717075447775e-03;
//		xcrref[2] = 0.2778153350936e-03;
//		xcrref[3] = 0.2887475409984e-03;
//		xcrref[4] = 0.3143611161242e-02;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of solution error.
//		//---------------------------------------------------------------------
//		xceref[0] = 0.7542088599534e-04;
//		xceref[1] = 0.6512852253086e-05;
//		xceref[2] = 0.1049092285688e-04;
//		xceref[3] = 0.1128838671535e-04;
//		xceref[4] = 0.1212845639773e-03;
//	} else if (nx == 64 && ny == 64 && nz == 64 && niter == 400) {
//		//---------------------------------------------------------------------
//		//    reference data for 64X64X64 grids after 400 time steps, with DT = 1.5d-03
//		//---------------------------------------------------------------------
//		*verifyclass = 'A';
//		dtref = 1.5e-3;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of residual.
//		//---------------------------------------------------------------------
//		xcrref[0] = 2.4799822399300195e0;
//		xcrref[1] = 1.1276337964368832e0;
//		xcrref[2] = 1.5028977888770491e0;
//		xcrref[3] = 1.4217816211695179e0;
//		xcrref[4] = 2.1292113035138280e0;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of solution error.
//		//---------------------------------------------------------------------
//		xceref[0] = 1.0900140297820550e-04;
//		xceref[1] = 3.7343951769282091e-05;
//		xceref[2] = 5.0092785406541633e-05;
//		xceref[3] = 4.7671093939528255e-05;
//		xceref[4] = 1.3621613399213001e-04;
//	} else if (nx == 102 && ny == 102 && nz == 102 && niter == 400) {
//		//---------------------------------------------------------------------
//		//    reference data for 102X102X102 grids after 400 time steps,
//		//    with DT = 1.0d-03
//		//---------------------------------------------------------------------
//		*verifyclass = 'B';
//		dtref = 1.0e-3;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of residual.
//		//---------------------------------------------------------------------
//		xcrref[0] = 0.6903293579998e+02;
//		xcrref[1] = 0.3095134488084e+02;
//		xcrref[2] = 0.4103336647017e+02;
//		xcrref[3] = 0.3864769009604e+02;
//		xcrref[4] = 0.5643482272596e+02;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of solution error.
//		//---------------------------------------------------------------------
//		xceref[0] = 0.9810006190188e-02;
//		xceref[1] = 0.1022827905670e-02;
//		xceref[2] = 0.1720597911692e-02;
//		xceref[3] = 0.1694479428231e-02;
//		xceref[4] = 0.1847456263981e-01;
//	} else if (nx == 162 && ny == 162 && nz == 162 && niter == 400) {
//		//---------------------------------------------------------------------
//		//    reference data for 162X162X162 grids after 400 time steps,
//		//    with DT = 0.67d-03
//		//---------------------------------------------------------------------
//		*verifyclass = 'C';
//		dtref = 0.67e-3;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of residual.
//		//---------------------------------------------------------------------
//		xcrref[0] = 0.5881691581829e+03;
//		xcrref[1] = 0.2454417603569e+03;
//		xcrref[2] = 0.3293829191851e+03;
//		xcrref[3] = 0.3081924971891e+03;
//		xcrref[4] = 0.4597223799176e+03;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of solution error.
//		//---------------------------------------------------------------------
//		xceref.replace('double', 'gdd_real')[0] = 0.2598120500183e+00;
//		xceref[1] = 0.2590888922315e-01;
//		xceref[2] = 0.5132886416320e-01;
//		xceref[3] = 0.4806073419454e-01;
//		xceref[4] = 0.5483377491301e+00;
//	} else if (nx == 408 && ny == 408 && nz == 408 && niter == 500) {
//		//---------------------------------------------------------------------
//		//    reference data for 408X408X408 grids after 500 time steps,
//		//    with DT = 0.3d-03
//		//---------------------------------------------------------------------
//		*verifyclass = 'D';
//		dtref = 0.30e-3;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of residual.
//		//---------------------------------------------------------------------
//		xcrref[0] = 0.1044696216887e+05;
//		xcrref[1] = 0.3204427762578e+04;
//		xcrref[2] = 0.4648680733032e+04;
//		xcrref[3] = 0.4238923283697e+04;
//		xcrref[4] = 0.7588412036136e+04;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of solution error.
//		//---------------------------------------------------------------------
//		xceref[0] = 0.5089471423669e+01;
//		xceref[1] = 0.5323514855894e+00;
//		xceref[2] = 0.1187051008971e+01;
//		xceref[3] = 0.1083734951938e+01;
//		xceref[4] = 0.1164108338568e+02;
//	} else if (nx == 1020 && ny == 1020 && nz == 1020 && niter == 500) {
//		//---------------------------------------------------------------------
//		//    reference data for 1020X1020X1020 grids after 500 time steps,
//		//    with DT = 0.1d-03
//		//---------------------------------------------------------------------
//		*verifyclass = 'E';
//		dtref = 0.10e-3;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of residual.
//		//---------------------------------------------------------------------
//		xcrref[0] = 0.6255387422609e+05;
//		xcrref[1] = 0.1495317020012e+05;
//		xcrref[2] = 0.2347595750586e+05;
//		xcrref[3] = 0.2091099783534e+05;
//		xcrref[4] = 0.4770412841218e+05;
//
//		//---------------------------------------------------------------------
//		//    Reference values of RMS-norms of solution error.
//		//---------------------------------------------------------------------
//		xceref[0] = 0.6742735164909e+02;
//		xceref[1] = 0.5390656036938e+01;
//		xceref[2] = 0.1680647196477e+02;
//		xceref[3] = 0.1536963126457e+02;
//		xceref[4] = 0.1575330146156e+03;
//	} else verified = false;
//
//	//---------------------------------------------------------------------
//	//    verification test for residuals if gridsize is one of 
//	//    the defined grid sizes above (class .ne. 'U')
//	//---------------------------------------------------------------------
//
//	//---------------------------------------------------------------------
//	//    Compute the difference of solution values and the known reference values.
//	//---------------------------------------------------------------------
//	double xcrdif[5], xcedif[5];
//	for (int m = 0; m < 5; m++) {
//		xcrdif[m] = fabs((xcr[m]-xcrref[m])/xcrref[m]);
//		xcedif[m] = fabs((xce[m]-xceref[m])/xceref[m]);
//	}
//
//	//---------------------------------------------------------------------
//	//    Output the comparison of computed results to known cases.
//	//---------------------------------------------------------------------
//	if (*verifyclass != 'U') {
//		printf(" Verification being performed for class %c\n", *verifyclass);
//		printf(" accuracy setting for epsilon = %20.13E\n", epsilon);
//		if (!verified) {
//			*verifyclass = 'U';
//			printf(" DT does not match the reference value of %15.8E\n", dtref);
//		}
//	} else printf(" Unknown class\n");
//
//	if (*verifyclass != 'U') printf(" Comparison of RMS-norms of residual\n");
//	else printf(" RMS-norms of residual\n");
//
//	for (int m = 0; m < 5; m++) {
//		if (*verifyclass == 'U') printf("          %2d%20.13E\n", m+1, xcr[m]);
//		else if (xcrdif[m] <= epsilon) printf("          %2d%20.13E%20.13E%20.13E\n", m+1, xcr[m], xcrref[m], xcrdif[m]);
//		else {
//			verified = false;
//			printf(" FAILURE: %2d%20.13E%20.13E%20.13E\n", m+1, xcr[m], xcrref[m], xcrdif[m]);
//		}
//	}
//
//	if (*verifyclass != 'U') printf(" Comparison of RMS-norms of solution error\n");
//	else printf(" RMS-norms of solution error\n");
//
//	for (int m = 0; m < 5; m++) {
//		if (*verifyclass == 'U') printf("          %2d%20.13E\n", m+1, xce[m]);
//		else if (xcedif[m] <= epsilon) printf("          %2d%20.13E%20.13E%20.13E\n", m+1, xce[m], xceref[m], xcedif[m]);
//		else {
//			verified = false;
//			printf(" FAILURE: %2d%20.13E%20.13E%20.13E\n", m+1, xce[m], xceref[m], xcedif[m]);
//		}
//	}
//
//	if (*verifyclass == 'U') {
//		printf(" No reference values provided\n");
//		printf(" No verification performed\n");
//	} else {
//		if (verified) printf(" Verification Successful\n");
//		else printf(" Verification failed\n");
//	}
//
//	return verified;
//}
//---------------------------------------------------------------------
//      Read input file (if it exists), else take
//      defaults from parameters
//---------------------------------------------------------------------
void read_input(char benchclass, double* dd_td, int* nx, int* ny, int* nz, int* niter) {
	FILE *file;

	if ((file = fopen("inputsp.data", "rt")) != NULL) {
		char line[1024];
		printf(" Reading from input file inputsp.data\n");
		
		fgets(line, sizeof(line)-1, file);
		sscanf(line, "%i", niter);
		fgets(line, sizeof(line)-1, file);
		sscanf(line, "%lf", dd_td);
		fgets(line, sizeof(line)-1, file);
		sscanf(line, "%i %i %i", nx, ny, nz);
		fclose(file);
	} else {
//		printf(" No input file inputsp.data. Using compiled defaults\n");
		int problem_size;
		switch (benchclass) {
			case 's':
			case 'S': problem_size = 12; *dd_td = 0.015; *niter = 100; break;
			case 'w':
			case 'W': problem_size = 36; *dd_td = 0.0015; *niter = 400; break;
			case 'a':
			case 'A': problem_size = 64; *dd_td = 0.0015; *niter = 400; break;
			case 'b':
			case 'B': problem_size = 102; *dd_td = 0.001; *niter = 400; break;
			case 'c':
			case 'C': problem_size = 162; *dd_td = 0.00067; *niter = 400; break;
			case 'd':
			case 'D': problem_size = 408; *dd_td = 0.00030; *niter = 500; break;
			case 'e':
			case 'E': problem_size = 1020; *dd_td = 0.0001; *niter = 500; break;
			default: printf("setparams: Internal error: invalid class %c\n", benchclass); exit(EXIT_FAILURE);
		}
		*nx = *ny = *nz = problem_size;
	}

	printf("\n\n NAS Parallel Benchmarks (NPB3.3-CUDA) - SP Benchmark\n\n");
	printf(" Size: %4dx%4dx%4d\n", *nx, *ny, *nz);
	printf(" Iterations: %4d    dt_d: %10.6F\n", *niter, *dd_td);
	printf("\n");
}




int main(int argc, char **argv) {
	char benchclass = argc > 1 ? argv[1][0] : 'S';

  struct timeval start_t;
  struct timeval end_t;

	int niter;
	int nx, ny, nz;
	double dd_d;
	
	double *u, *forcing, *rhs, *rho_i, *us, *vs, *ws, *qs, *speed, *square, *lhs, *rhstmp;
  //double* rmsbuf;
	//double xce[5], xcr[5];

	char CUDAname[256];
	int CUDAmp, CUDAclock, CUDAmemclock, CUDAl2cache;
	size_t CUDAmem;

	//---------------------------------------------------------------------
	//   read input data
	//---------------------------------------------------------------------
	read_input(benchclass, &dd_d, &nx, &ny, &nz, &niter);

	//---------------------------------------------------------------------
	//   allocate CUDA device memory
	//---------------------------------------------------------------------
	int gridsize = nx*ny*nz;
	int facesize = max(max(nx*ny, nx*nz), ny*nz);

  gettimeofday(&start_t, NULL);

	HANDLE_ERROR(hipMalloc((void **)&u, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&forcing, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rhs, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rho_i, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&us, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&vs, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&ws, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&qs, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&speed, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&square, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&lhs, 9*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rhstmp, 5*gridsize*sizeof(double)));
	//HANDLE_ERROR(cudaMalloc((void **)&rmsbuf, 5*facesize*sizeof(double)));

	double ce_d[13][5];
	ce_d[0][0] = (double)2.0;
	ce_d[1][0] = (double)0.0;
	ce_d[2][0] = (double)0.0;
	ce_d[3][0] = (double)4.0;
	ce_d[4][0] = (double)5.0;
	ce_d[5][0] = (double)3.0;
	ce_d[6][0] = (double)0.5;
	ce_d[7][0] = (double)0.02;
	ce_d[8][0] = (double)0.01;
	ce_d[9][0] = (double)0.03;
	ce_d[10][0] = (double)0.5;
	ce_d[11][0] = (double)0.4;
	ce_d[12][0] = (double)0.3;

	ce_d[0][1] = (double)1.0;
	ce_d[1][1] = (double)0.0;
	ce_d[2][1] = (double)0.0;
	ce_d[3][1] = (double)0.0;
	ce_d[4][1] = (double)1.0;
	ce_d[5][1] = (double)2.0;
	ce_d[6][1] = (double)3.0;
	ce_d[7][1] = (double)0.01;
	ce_d[8][1] = (double)0.03;
	ce_d[9][1] = (double)0.02;
	ce_d[10][1] = (double)0.4;
	ce_d[11][1] = (double)0.3;
	ce_d[12][1] = (double)0.5;

	ce_d[0][2] = (double)2.0;
	ce_d[1][2] = (double)2.0;
	ce_d[2][2] = (double)0.0;
	ce_d[3][2] = (double)0.0;
	ce_d[4][2] = (double)0.0;
	ce_d[5][2] = (double)2.0;
	ce_d[6][2] = (double)3.0;
	ce_d[7][2] = (double)0.04;
	ce_d[8][2] = (double)0.03;
	ce_d[9][2] = (double)0.05;
	ce_d[10][2] = (double)0.3;
	ce_d[11][2] = (double)0.5;
	ce_d[12][2] = (double)0.4;

	ce_d[0][3] = (double)2.0;
	ce_d[1][3] = (double)2.0;
	ce_d[2][3] = (double)0.0;
	ce_d[3][3] = (double)0.0;
	ce_d[4][3] = (double)0.0;
	ce_d[5][3] = (double)2.0;
	ce_d[6][3] = (double)3.0;
	ce_d[7][3] = (double)0.03;
	ce_d[8][3] = (double)0.05;
	ce_d[9][3] = (double)0.04;
	ce_d[10][3] = (double)0.2;
	ce_d[11][3] = (double)0.1;
	ce_d[12][3] = (double)0.3;

	ce_d[0][4] = (double)5.0;
	ce_d[1][4] = (double)4.0;
	ce_d[2][4] = (double)3.0;
	ce_d[3][4] = (double)2.0;
	ce_d[4][4] = (double)0.1;
	ce_d[5][4] = (double)0.4;
	ce_d[6][4] = (double)0.3;
	ce_d[7][4] = (double)0.05;
	ce_d[8][4] = (double)0.04;
	ce_d[9][4] = (double)0.03;
	ce_d[10][4] = (double)0.1;
	ce_d[11][4] = (double)0.3;
	ce_d[12][4] = (double)0.2;

	double bt_d = sqrt(0.5);

	double dnxm1_d = 1.0/((double)nx-1.0);
	double dnym1_d = 1.0/((double)ny-1.0);
	double dnzm1_d = 1.0/((double)nz-1.0);

	double tx1_d = 1.0 / (dnxm1_d * dnxm1_d);
	double tx2_d = 1.0 / (2.0 * dnxm1_d);
	double tx3_d = 1.0 / dnxm1_d;

	double ty1_d = 1.0 / (dnym1_d * dnym1_d);
	double ty2_d = 1.0 / (2.0 * dnym1_d);
	double ty3_d = 1.0 / dnym1_d;
 
	double tz1_d = 1.0 / (dnzm1_d * dnzm1_d);
	double tz2_d = 1.0 / (2.0 * dnzm1_d);
	double tz3_d = 1.0 / dnzm1_d;

	double dttx1_d = dd_d*tx1_d;
	double dttx2_d = dd_d*tx2_d;
	double dtty1_d = dd_d*ty1_d;
	double dtty2_d = dd_d*ty2_d;
	double dttz1_d = dd_d*tz1_d;
	double dttz2_d = dd_d*tz2_d;

	double c2dttx1_d = 2.0*dttx1_d;
	double c2dtty1_d = 2.0*dtty1_d;
	double c2dttz1_d = 2.0*dttz1_d;

	double dtdssp_d = dd_d*dssp;

	double comz1_d  = dtdssp_d;
	double comz4_d  = 4.0*dtdssp_d;
	double comz5_d  = 5.0*dtdssp_d;
	double comz6_d  = 6.0*dtdssp_d;

	double c3c4tx3_d = c3c4*tx3_d;
	double c3c4ty3_d = c3c4*ty3_d;
	double c3c4tz3_d = c3c4*tz3_d;

	double dx1tx1_d = dx1*tx1_d;
	double dx2tx1_d = dx2*tx1_d;
	double dx3tx1_d = dx3*tx1_d;
	double dx4tx1_d = dx4*tx1_d;
	double dx5tx1_d = dx5*tx1_d;

	double dy1ty1_d = dy1*ty1_d;
	double dy2ty1_d = dy2*ty1_d;
	double dy3ty1_d = dy3*ty1_d;
	double dy4ty1_d = dy4*ty1_d;
	double dy5ty1_d = dy5*ty1_d;

	double dz1tz1_d = dz1*tz1_d;
	double dz2tz1_d = dz2*tz1_d;
	double dz3tz1_d = dz3*tz1_d;
	double dz4tz1_d = dz4*tz1_d;
	double dz5tz1_d = dz5*tz1_d;

	double xxcon1_d = c3c4tx3_d*con43*tx3_d;
	double xxcon2_d = c3c4tx3_d*tx3_d;
	double xxcon3_d = c3c4tx3_d*conz1*tx3_d;
	double xxcon4_d = c3c4tx3_d*con16*tx3_d;
	double xxcon5_d = c3c4tx3_d*c1c5*tx3_d;

	double yycon1_d = c3c4ty3_d*con43*ty3_d;
	double yycon2_d = c3c4ty3_d*ty3_d;
	double yycon3_d = c3c4ty3_d*conz1*ty3_d;
	double yycon4_d = c3c4ty3_d*con16*ty3_d;
	double yycon5_d = c3c4ty3_d*c1c5*ty3_d;

	double zzcon1_d = c3c4tz3_d*con43*tz3_d;
	double zzcon2_d = c3c4tz3_d*tz3_d;
	double zzcon3_d = c3c4tz3_d*conz1*tz3_d;
	double zzcon4_d = c3c4tz3_d*con16*tz3_d;
	double zzcon5_d = c3c4tz3_d*c1c5*tz3_d;

	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&ce), &ce_d, 13*5*sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&bt), &bt_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dnxm1), &dnxm1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dnym1), &dnym1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dnzm1), &dnzm1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&tx1), &tx1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&tx2), &tx2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&tx3), &tx3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&ty1), &ty1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&ty2), &ty2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&ty3), &ty3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&tz1), &tz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&tz2), &tz2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&tz3), &tz3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dttx1), &dttx1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dttx2), &dttx2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dtty1), &dtty1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dtty2), &dtty2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dttz1), &dttz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dttz2), &dttz2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&c2dttx1), &c2dttx1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&c2dtty1), &c2dtty1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&c2dttz1), &c2dttz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dt), &dd_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dtdssp), &dtdssp_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&comz1), &comz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&comz4), &comz4_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&comz5), &comz5_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&comz6), &comz6_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&c3c4tx3), &c3c4tx3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&c3c4ty3), &c3c4ty3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&c3c4tz3), &c3c4tz3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dx1tx1), &dx1tx1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dx2tx1), &dx2tx1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dx3tx1), &dx3tx1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dx4tx1), &dx4tx1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dx5tx1), &dx5tx1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dy1ty1), &dy1ty1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dy2ty1), &dy2ty1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dy3ty1), &dy3ty1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dy4ty1), &dy4ty1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dy5ty1), &dy5ty1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dz1tz1), &dz1tz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dz2tz1), &dz2tz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dz3tz1), &dz3tz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dz4tz1), &dz4tz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&dz5tz1), &dz5tz1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&xxcon1), &xxcon1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&xxcon2), &xxcon2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&xxcon3), &xxcon3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&xxcon4), &xxcon4_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&xxcon5), &xxcon5_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&yycon1), &yycon1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&yycon2), &yycon2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&yycon3), &yycon3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&yycon4), &yycon4_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&yycon5), &yycon5_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&zzcon1), &zzcon1_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&zzcon2), &zzcon2_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&zzcon3), &zzcon3_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&zzcon4), &zzcon4_d, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol (HIP_SYMBOL(&zzcon5), &zzcon5_d, sizeof(double)));



	exact_rhs(forcing, nx, ny, nz);

	//sp->initialize();
  dim3 grid(nz,ny);
	initialize_kernel<<<grid,nx>>> (u, nx, ny, nz);

	//---------------------------------------------------------------------
	//      do one time step to touch all code, and reinitialize
	//---------------------------------------------------------------------
	adi(true,  nx,  ny,  nz,  niter,  rho_i,  us,  vs,  ws, 
       speed,  qs,  square,  rhs,  lhs,  forcing,  u,  rhstmp);
	//sp->initialize();
	initialize_kernel<<<grid,nx>>> (u, nx, ny, nz);

	//---------------------------------------------------------------------
	//   main time stepping loop
	//---------------------------------------------------------------------
	//sp->adi(false);
	adi(false,  nx,  ny,  nz,  niter,  rho_i,  us,  vs,  ws, 
       speed,  qs,  square,  rhs,  lhs,  forcing,  u,  rhstmp);
  gettimeofday(&end_t, NULL);

	std::cout  << "time: "<<((end_t.tv_sec-start_t.tv_sec)+(end_t.tv_usec-start_t.tv_usec)*1e-6) << std::endl;
	//std::cout  << (sdkGetAverageTimerValue(&timer)/1000.0)  / iterations << " seconds per iteration" << std::endl;
	//---------------------------------------------------------------------
	//   verification test
	//---------------------------------------------------------------------
	//char verifyclass;
  //bool verified;
	//verified = verify(&verifyclass,  rmsbuf,  u,  rho_i,  us,  vs,  ws,  qs, square,  rhs,  forcing ,  speed, xce, xcr, dd_d, nx,  ny,  nz, niter); 
	//sp->print_results(verified, verifyclass);

	//---------------------------------------------------------------------
	//      More timers
	//---------------------------------------------------------------------
	//sp->print_timers();

	//delete sp;
	return EXIT_SUCCESS;
}
