#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <getopt.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <mpfr.h>
#include <iostream>

using namespace std;

#ifdef RD_WG_SIZE_0_0
        #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE RD_WG_SIZE
#else
        #define BLOCK_SIZE 16
#endif

#define MIN(i,j) ((i)<(j) ? (i) : (j))

#define GET_RAND_FP ( (double)rand() /   \
                     ((double)(RAND_MAX)+(double)(1)) )


static struct option long_options[] = {
  /* name, has_arg, flag, val */
  {"input", 1, NULL, 'i'},
  {"size", 1, NULL, 's'},
  {"verify", 0, NULL, 'v'},
  {0,0,0,0}
};

__global__ void 
lud_diagonal(double *m, int matrix_dim, int offset)
{
  int i,j;
  __shared__ double shadow[BLOCK_SIZE][BLOCK_SIZE];

  int array_offset = offset*matrix_dim+offset;
  for(i=0; i < BLOCK_SIZE; i++){
    shadow[i][threadIdx.x]=m[array_offset+threadIdx.x];
    array_offset += matrix_dim;
  }
  __syncthreads();
  for(i=0; i < BLOCK_SIZE-1; i++) {

    if (threadIdx.x>i){
      for(j=0; j < i; j++)
        shadow[threadIdx.x][i] -= shadow[threadIdx.x][j]*shadow[j][i];
      shadow[threadIdx.x][i] /= shadow[i][i];
    }

    __syncthreads();
    if (threadIdx.x>i){

      for(j=0; j < i+1; j++)
        shadow[i+1][threadIdx.x] -= shadow[i+1][j]*shadow[j][threadIdx.x];
    }
    __syncthreads();
  }

   array_offset = (offset+1)*matrix_dim+offset;
  for(i=1; i < BLOCK_SIZE; i++){
    m[array_offset+threadIdx.x]=shadow[i][threadIdx.x];
    array_offset += matrix_dim;
  }
}

__global__ void
lud_perimeter(double *m, int matrix_dim, int offset)
{
  __shared__ double dia[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i,j, array_offset;
  int idx;

  if (threadIdx.x < BLOCK_SIZE) {
    idx = threadIdx.x;
    
    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE/2; i++){
      dia[i][idx]=m[array_offset+idx];
      array_offset += matrix_dim;
    }
    
    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      peri_row[i][idx]=m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx];
      array_offset += matrix_dim;
    }

  } else {
    idx = threadIdx.x-BLOCK_SIZE;
    
    array_offset = (offset+BLOCK_SIZE/2)*matrix_dim+offset;
    for (i=BLOCK_SIZE/2; i < BLOCK_SIZE; i++){
      dia[i][idx]=m[array_offset+idx];
      array_offset += matrix_dim;
    }
    
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      peri_col[i][idx] = m[array_offset+idx];
      array_offset += matrix_dim;
    }
  
  }
  __syncthreads();

  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    for(i=1; i < BLOCK_SIZE; i++){
      for (j=0; j < i; j++)
        peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    for(i=0; i < BLOCK_SIZE; i++){
      for(j=0; j < i; j++)
        peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
      peri_col[idx][i] /= dia[i][i];
    }
  }

  __syncthreads();
    
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    array_offset = (offset+1)*matrix_dim+offset;
    for(i=1; i < BLOCK_SIZE; i++){
      m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
      array_offset += matrix_dim;
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for(i=0; i < BLOCK_SIZE; i++){
      m[array_offset+idx] =  peri_col[i][idx];
      array_offset += matrix_dim;
    }
  }

}

__global__ void
lud_internal(double *m, int matrix_dim, int offset)
{
  __shared__ double peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i;
  double sum;

  int global_row_id = offset + (blockIdx.y+1)*BLOCK_SIZE;
  int global_col_id = offset + (blockIdx.x+1)*BLOCK_SIZE;

  peri_row[threadIdx.y][threadIdx.x] = m[(offset+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x];
  peri_col[threadIdx.y][threadIdx.x] = m[(global_row_id+threadIdx.y)*matrix_dim+offset+threadIdx.x];

  __syncthreads();

  sum = 0;
  for (i=0; i < BLOCK_SIZE; i++)
    sum += peri_col[threadIdx.y][i] * peri_row[i][threadIdx.x];
  m[(global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x] -= sum;


}

void create_matrix(double *trix, int size){
  int i,j;
  double lamda = -0.001;
  double cof[2*size-1];
  double coe_i =0.0;

  for (i=0; i < size; i++)
    {
      coe_i = 10*exp(lamda*i); 
      j=size-1+i;     
      cof[j]=coe_i;
      j=size-1-i;     
      cof[j]=coe_i;
    }

  for (i=0; i < size; i++) {
      for (j=0; j < size; j++) {
	      trix[i*size+j]=cof[size-1-i+j];
      }
  }
}

int main ( int argc, char *argv[] ) {
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

  int matrix_dim = 32; /* default matrix_dim */
  int opt, option_index=0;
  int i, j;
  const char *input_file = NULL;
  double *d_m;
  struct timeval start_t;
  struct timeval end_t;
  struct timeval skt_t;
  struct timeval ske_t;

  while ((opt = getopt_long(argc, argv, "::vs:i:", 
                            long_options, &option_index)) != -1 ) {
    switch(opt){
    case 'i':
      input_file = optarg;
      break;
    case 's':
      matrix_dim = atoi(optarg);
      printf("Generate input matrix internally, size =%d\n", matrix_dim);
      break;
    case '?':
      fprintf(stderr, "invalid option\n");
      break;
    case ':':
      fprintf(stderr, "missing argument\n");
      break;
    default:
      fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n",
	      argv[0]);
      exit(EXIT_FAILURE);
    }
  }
  
  if ( (optind < argc) || (optind == 1)) {
    fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
    exit(EXIT_FAILURE);
  }

  double* fv = new double[matrix_dim*matrix_dim];

  if (input_file) {
    printf("Reading matrix from file %s\n", input_file);
    FILE *fp = NULL;
    float input;
     fp = fopen(input_file, "rb");
     fscanf(fp, "%d\n", &matrix_dim);
     //m = (double*) malloc(sizeof(double)*size*size);
     for (i=0; i < matrix_dim; i++) {
         for (j=0; j < matrix_dim; j++) {
             fscanf(fp, "%f ", &input);
             fv[i*matrix_dim+j] = (double)input; 
         }
     }
     fclose(fp);
     printf("matrix dim: %d\n", matrix_dim);
  } 
  else if (matrix_dim) {
    printf("Creating matrix internally size=%d\n", matrix_dim);
    create_matrix(fv, matrix_dim);
  }
  else {
    printf("No input file specified!\n");
    exit(EXIT_FAILURE);
  }

  hipMalloc((void**)&d_m, matrix_dim*matrix_dim*sizeof(double));

  gettimeofday(&start_t,0L);
  /* beginning of timing point */
  hipMemcpy(d_m, fv, matrix_dim*matrix_dim*sizeof(double), 
	     hipMemcpyHostToDevice);

  //lud_cuda(d_m, matrix_dim);
  gettimeofday(&skt_t,0L);
    i = 0;
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //double *m_debug = (double*)malloc(matrix_dim*matrix_dim*sizeof(double));
    for (i=0; i < matrix_dim-BLOCK_SIZE; i += BLOCK_SIZE) {
        lud_diagonal<<<1, BLOCK_SIZE>>>(d_m, matrix_dim, i);
        lud_perimeter<<<(matrix_dim-i)/BLOCK_SIZE-1, BLOCK_SIZE*2>>>(d_m, matrix_dim, i);
        dim3 dimGrid((matrix_dim-i)/BLOCK_SIZE-1, (matrix_dim-i)/BLOCK_SIZE-1);
        lud_internal<<<dimGrid, dimBlock>>>(d_m, matrix_dim, i); 
    }
    lud_diagonal<<<1,BLOCK_SIZE>>>(d_m, matrix_dim, i);
	  hipDeviceSynchronize();
  gettimeofday(&ske_t,0L);

  hipMemcpy(fv, d_m, matrix_dim*matrix_dim*sizeof(double), 
	     hipMemcpyDeviceToHost);

  /* end of timing point */

  gettimeofday(&end_t,0L);

#if 1
  mpf_t val_x, val_y, val_in, err;
  mpf_init2(val_x, 128);
  mpf_init2(val_y, 128);
  mpf_init2(val_in, 128);
  mpf_init2(err, 128);
  FILE *infile = fopen("m_ref.txt", "r");
  for (i=0; i<matrix_dim*matrix_dim; i++) {
    gmp_fscanf(infile, "%Fe\n", val_in);
    mpf_set_d(val_x, fv[i]);
    mpf_sub(val_y, val_x, val_in);
    mpf_abs(val_x, val_y);
    mpf_div(val_x, val_x, val_in);
    if (i==0)
      mpf_set(err, val_x);
    else
      mpf_add(err, err, val_x);
  }
  mpf_div_ui(err, err, matrix_dim*matrix_dim);
  gmp_printf("error: %.80Ff\n", err);
#else
  mpf_t val_x;
  mpf_init2(val_x, 128);
  FILE *outfile = fopen("m_ref.txt", "w");
  for (int i=0; i<matrix_dim*matrix_dim; i++) {
    mpf_set_d(val_x, fv[i]);
    gmp_fprintf(outfile, "%.80Ff\n", val_x);
  }

#endif

  std::cout << "time: " << ((end_t.tv_sec + end_t.tv_usec*1e-6) - (start_t.tv_sec + start_t.tv_usec*1e-6)) << "\n";
  std::cout <<"kernel: " << ((ske_t . tv_sec - skt_t . tv_sec) + (ske_t . tv_usec - skt_t . tv_usec) * 1e-6) << endl;
  hipFree(d_m);
  free(fv);

  return EXIT_SUCCESS;
}				/* ----------  end of function main  ---------- */
