#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  lud.cu
 *
 *    Description:  The main wrapper for the suite
 *
 *        Version:  1.0
 *        Created:  10/22/2009 08:40:34 PM
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Liang Wang (lw2aw), lw2aw@virginia.edu
 *        Company:  CS@UVa
 *
 * =====================================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <getopt.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <mpfr.h>

#ifdef RD_WG_SIZE_0_0
        #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE RD_WG_SIZE
#else
        #define BLOCK_SIZE 16
#endif

#define MIN(i,j) ((i)<(j) ? (i) : (j))

#define GET_RAND_FP ( (double)rand() /   \
                     ((double)(RAND_MAX)+(double)(1)) )

static int do_verify = 0;

static struct option long_options[] = {
  /* name, has_arg, flag, val */
  {"input", 1, NULL, 'i'},
  {"size", 1, NULL, 's'},
  {"verify", 0, NULL, 'v'},
  {0,0,0,0}
};

__global__ void 
lud_diagonal(double *m, int matrix_dim, int offset)
{
  int i,j;
  __shared__ double shadow[BLOCK_SIZE][BLOCK_SIZE];

  int array_offset = offset*matrix_dim+offset;
  for(i=0; i < BLOCK_SIZE; i++){
    shadow[i][threadIdx.x]=m[array_offset+threadIdx.x];
    array_offset += matrix_dim;
  }
  __syncthreads();
  for(i=0; i < BLOCK_SIZE-1; i++) {

    if (threadIdx.x>i){
      for(j=0; j < i; j++)
        shadow[threadIdx.x][i] -= shadow[threadIdx.x][j]*shadow[j][i];
      shadow[threadIdx.x][i] /= shadow[i][i];
    }

    __syncthreads();
    if (threadIdx.x>i){

      for(j=0; j < i+1; j++)
        shadow[i+1][threadIdx.x] -= shadow[i+1][j]*shadow[j][threadIdx.x];
    }
    __syncthreads();
  }

  /* 
     The first row is not modified, it
     is no need to write it back to the
     global memory

   */
  array_offset = (offset+1)*matrix_dim+offset;
  for(i=1; i < BLOCK_SIZE; i++){
    m[array_offset+threadIdx.x]=shadow[i][threadIdx.x];
    array_offset += matrix_dim;
  }
}

__global__ void
lud_perimeter(double *m, int matrix_dim, int offset)
{
  __shared__ double dia[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i,j, array_offset;
  int idx;

  if (threadIdx.x < BLOCK_SIZE) {
    idx = threadIdx.x;
    
    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE/2; i++){
      dia[i][idx]=m[array_offset+idx];
      array_offset += matrix_dim;
    }
    
    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      peri_row[i][idx]=m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx];
      array_offset += matrix_dim;
    }

  } else {
    idx = threadIdx.x-BLOCK_SIZE;
    
    array_offset = (offset+BLOCK_SIZE/2)*matrix_dim+offset;
    for (i=BLOCK_SIZE/2; i < BLOCK_SIZE; i++){
      dia[i][idx]=m[array_offset+idx];
      array_offset += matrix_dim;
    }
    
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      peri_col[i][idx] = m[array_offset+idx];
      array_offset += matrix_dim;
    }
  
  }
  __syncthreads();

  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    for(i=1; i < BLOCK_SIZE; i++){
      for (j=0; j < i; j++)
        peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    for(i=0; i < BLOCK_SIZE; i++){
      for(j=0; j < i; j++)
        peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
      peri_col[idx][i] /= dia[i][i];
    }
  }

  __syncthreads();
    
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    array_offset = (offset+1)*matrix_dim+offset;
    for(i=1; i < BLOCK_SIZE; i++){
      m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
      array_offset += matrix_dim;
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for(i=0; i < BLOCK_SIZE; i++){
      m[array_offset+idx] =  peri_col[i][idx];
      array_offset += matrix_dim;
    }
  }

}

__global__ void
lud_internal(double *m, int matrix_dim, int offset)
{
  __shared__ double peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i;
  double sum;

  int global_row_id = offset + (blockIdx.y+1)*BLOCK_SIZE;
  int global_col_id = offset + (blockIdx.x+1)*BLOCK_SIZE;

  peri_row[threadIdx.y][threadIdx.x] = m[(offset+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x];
  peri_col[threadIdx.y][threadIdx.x] = m[(global_row_id+threadIdx.y)*matrix_dim+offset+threadIdx.x];

  __syncthreads();

  sum = 0;
  for (i=0; i < BLOCK_SIZE; i++)
    sum += peri_col[threadIdx.y][i] * peri_row[i][threadIdx.x];
  m[(global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x] -= sum;


}

void create_matrix_from_random(double **mp, int size){
  double *l, *u, *m;
  int i,j,k;

  srand(time(NULL));

  l = (double*)malloc(size*size*sizeof(double));

  u = (double*)malloc(size*size*sizeof(double));

  for (i = 0; i < size; i++) {
      for (j=0; j < size; j++) {
          if (i>j) {
              l[i*size+j] = GET_RAND_FP;
          } else if (i == j) {
              l[i*size+j] = 1;
          } else {
              l[i*size+j] = 0;
          }
      }
  }

  for (j=0; j < size; j++) {
      for (i=0; i < size; i++) {
          if (i>j) {
              u[j*size+i] = 0;
          }else {
              u[j*size+i] = GET_RAND_FP; 
          }
      }
  }

  for (i=0; i < size; i++) {
      for (j=0; j < size; j++) {
          for (k=0; k <= MIN(i,j); k++)
            m[i*size+j] = l[i*size+k] * u[j*size+k];
      }
  }

  free(l);
  free(u);

  *mp = m;
}

void create_matrix(double **mp, int size){
  double *m;
  int i,j;
  double lamda = -0.001;
  double coe[2*size-1];
  double coe_i =0.0;

  for (i=0; i < size; i++)
    {
      coe_i = 10*exp(lamda*i); 
      j=size-1+i;     
      coe[j]=coe_i;
      j=size-1-i;     
      coe[j]=coe_i;
    }

  m = (double*) malloc(sizeof(double)*size*size);

  for (i=0; i < size; i++) {
      for (j=0; j < size; j++) {
	m[i*size+j]=coe[size-1-i+j];
      }
  }

  *mp = m;
}

int main ( int argc, char *argv[] ) {
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

  int matrix_dim = 32; /* default matrix_dim */
  int opt, option_index=0;
  int i, j;
  const char *input_file = NULL;
  double *m, *d_m;

  while ((opt = getopt_long(argc, argv, "::vs:i:", 
                            long_options, &option_index)) != -1 ) {
    switch(opt){
    case 'i':
      input_file = optarg;
      break;
    case 'v':
      do_verify = 1;
      break;
    case 's':
      matrix_dim = atoi(optarg);
      printf("Generate input matrix internally, size =%d\n", matrix_dim);
      // fprintf(stderr, "Currently not supported, use -i instead\n");
      // fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
      // exit(EXIT_FAILURE);
      break;
    case '?':
      fprintf(stderr, "invalid option\n");
      break;
    case ':':
      fprintf(stderr, "missing argument\n");
      break;
    default:
      fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n",
	      argv[0]);
      exit(EXIT_FAILURE);
    }
  }
  
  if ( (optind < argc) || (optind == 1)) {
    fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
    exit(EXIT_FAILURE);
  }

  if (input_file) {
    printf("Reading matrix from file %s\n", input_file);
    FILE *fp = NULL;
    float input;
     fp = fopen(input_file, "rb");
     fscanf(fp, "%d\n", &matrix_dim);
     m = new double[matrix_dim*matrix_dim];
     //m = (double*) malloc(sizeof(double)*size*size);
     for (i=0; i < matrix_dim; i++) {
         for (j=0; j < matrix_dim; j++) {
             fscanf(fp, "%f ", &input);
             *(m+i*matrix_dim+j) = (double)input; 
         }
     }
     fclose(fp);
     printf("matrix dim: %d\n", matrix_dim);
    //create_matrix_from_file(m, input_file, &matrix_dim);
  } 
  else if (matrix_dim) {
    printf("Creating matrix internally size=%d\n", matrix_dim);
    create_matrix(&m, matrix_dim);
  }
  else {
    printf("No input file specified!\n");
    exit(EXIT_FAILURE);
  }

  hipMalloc((void**)&d_m, 
             matrix_dim*matrix_dim*sizeof(double));

  struct timeval time_start;
  struct timeval time_end;
  gettimeofday(&time_start, NULL);	
  /* beginning of timing point */
  hipMemcpy(d_m, m, matrix_dim*matrix_dim*sizeof(double), 
	     hipMemcpyHostToDevice);

  //lud_cuda(d_m, matrix_dim);
    i = 0;
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //double *m_debug = (double*)malloc(matrix_dim*matrix_dim*sizeof(double));
    for (i=0; i < matrix_dim-BLOCK_SIZE; i += BLOCK_SIZE) {
        lud_diagonal<<<1, BLOCK_SIZE>>>(d_m, matrix_dim, i);
        lud_perimeter<<<(matrix_dim-i)/BLOCK_SIZE-1, BLOCK_SIZE*2>>>(d_m, matrix_dim, i);
        dim3 dimGrid((matrix_dim-i)/BLOCK_SIZE-1, (matrix_dim-i)/BLOCK_SIZE-1);
        lud_internal<<<dimGrid, dimBlock>>>(d_m, matrix_dim, i); 
    }
    lud_diagonal<<<1,BLOCK_SIZE>>>(d_m, matrix_dim, i);

  hipMemcpy(m, d_m, matrix_dim*matrix_dim*sizeof(double), 
	     hipMemcpyDeviceToHost);

  /* end of timing point */

  gettimeofday(&time_end, NULL);

  mpf_t val_x, val_y, val_in, err;
  mpf_init2(val_x, 128);
  mpf_init2(val_y, 128);
  mpf_init2(val_in, 128);
  mpf_init2(err, 128);
  FILE *infile = fopen("m_ref.txt", "r");
  for (i=0; i<matrix_dim*matrix_dim; i++) {
    gmp_fscanf(infile, "%Fe\n", val_in);
    mpf_set_d(val_x, m[i]);
    mpf_sub(val_y, val_x, val_in);
    mpf_abs(val_x, val_y);
    mpf_div(val_x, val_x, val_in);
    if (i==0)
      mpf_set(err, val_x);
    else
      mpf_add(err, err, val_x);
  }
  mpf_div_ui(err, err, matrix_dim*matrix_dim);
  gmp_printf("error: %.80Ff\n", err);

  double time_total = (time_end.tv_sec + time_end.tv_usec*1e-6) - (time_start.tv_sec + time_start.tv_usec*1e-6);
  printf("time: %f\n", time_total);
  //if (do_verify){
  //  printf("After LUD\n");
  //  // print_matrix(m, matrix_dim);
  //  printf(">>>Verify<<<<\n");
  //  lud_verify(mm, m, matrix_dim); 
  //  free(mm);
  //}

  hipFree(d_m);
  free(m);

  return EXIT_SUCCESS;
}				/* ----------  end of function main  ---------- */
