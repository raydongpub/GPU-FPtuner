#include "hip/hip_runtime.h"
//#include <hip/hip_runtime_api.h>

#include <helper_timer.h>

#include <mpfr.h>

#include <qd/dd_real.h>

#include "../../gpuprec/gqd/gqd.cu"

using namespace std;

void qd2gqd(dd_real* dd_data, gdd_real* gdd_data, const unsigned int numElement) {

    for (unsigned int i = 0; i < numElement; i++) {

        gdd_data[i].x = dd_data[i].x[0];

        gdd_data[i].y = dd_data[i].x[1];

    }

}

void gqd2qd(gdd_real* gdd_data, dd_real* dd_data, const unsigned int numElement) {

    for (unsigned int i = 0; i < numElement; i++) {

        dd_data[i].x[0] = gdd_data[i].x;

        dd_data[i].x[1] = gdd_data[i].y;

    }

}

void qd2gqd2(dd_real dd_data[][5], gdd_real gdd_data[][5], int d1, int d2, int numElement) {

    for (unsigned int i = 0; i < d1; i++) {

      for (unsigned int j = 0; j < d2; j++) {

        gdd_data[i][j].x = dd_data[i][j].x[0];

        gdd_data[i][j].y = dd_data[i][j].x[1];

      }

    }

}

#include <stdio.h>					

#include <stdlib.h>					

#include <stdbool.h>				

#include <iostream>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>

#include <sys/time.h>

#include <mpfr.h>

#include <math.h>

#include <string.h>

#define NUMBER_PAR_PER_BOX 100							

#ifdef RD_WG_SIZE_0_0

        #define NUMBER_THREADS RD_WG_SIZE_0_0

#elif defined(RD_WG_SIZE_0)

        #define NUMBER_THREADS RD_WG_SIZE_0

#elif defined(RD_WG_SIZE)

        #define NUMBER_THREADS RD_WG_SIZE

#else

        #define NUMBER_THREADS 128

#endif

#define DOT(A,B) ((A.x)*(B.x)+(A.y)*(B.y)+(A.z)*(B.z))	// STABLE

using namespace std;

typedef struct nei_str {

// neighbor box

int x;

int y;

int z;

int number;

long offset;}nei_str;

typedef struct box_str {

// home box

int x;

int y;

int z;

int number;

long offset;

// neighbor boxes

int nn;

::nei_str nei[26];}box_str;

typedef struct dim_str {

// input arguments

int cur_arg;

int arch_arg;

int cores_arg;

int boxes1d_arg;

// system memory

long number_boxes;

long box_mem;

long space_elem;}dim_str;

int isInteger(char *str)

{

  if (( *str) == '\0') {

    return 0;

  }

  for (; ( *str) != '\0'; str++) {

    if (( *str) < 48 || ( *str) > 57) {

      return 0;

    }

  }

  return 1;

}

void checkCUDAError(const char *msg)

{

  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {

    printf("Cuda error: %s: %s.\n",msg,(hipGetErrorString(err)));

    fflush(0L);

    exit(1);

  }

}

__constant__ float dev_par;

__global__ void kernel_gpu_cuda(::dim_str d_dim_gpu,::box_str *d_box_gpu,float *d_rv_v,float *d_rv_x,float *d_rv_y,float *d_rv_z,float *d_qv_gpu,float *d_fv_v,float *d_fv_x,float *d_fv_y,float *d_fv_z)

{

  int bx = blockIdx . x;

  int tx = threadIdx . x;

  int wtx = tx;

  if (bx < d_dim_gpu . number_boxes) {

// parameters

    float a2 = 2.0 * dev_par * dev_par;

// home box

    int first_i;

    float *rA_v;

    float *rA_x;

    float *rA_y;

    float *rA_z;

    float *fA_v;

    float *fA_x;

    float *fA_y;

    float *fA_z;

    __shared__ float rA_shared_v[100];

    __shared__ float rA_shared_x[100];

    __shared__ float rA_shared_y[100];

    __shared__ float rA_shared_z[100];

// nei box

    int pointer;

    int k = 0;

    int first_j;

    float *rB_v;

    float *rB_x;

    float *rB_y;

    float *rB_z;

    float *qB;

    int j = 0;

    __shared__ float rB_shared_v[100];

    __shared__ float rB_shared_x[100];

    __shared__ float rB_shared_y[100];

    __shared__ float rB_shared_z[100];

    __shared__ float qb_shared[100];

// common

    float r2;

    float u2;

    float vij;

    float fs;

    float fxij;

    float fyij;

    float fzij;

    float s_x;

    float s_y;

    float s_z;

    first_i = d_box_gpu[bx] . offset;

    rA_v =( &d_rv_v[first_i]);

    rA_x =( &d_rv_x[first_i]);

    rA_y =( &d_rv_y[first_i]);

    rA_z =( &d_rv_z[first_i]);

    fA_v =( &d_fv_v[first_i]);

    fA_x =( &d_fv_x[first_i]);

    fA_y =( &d_fv_y[first_i]);

    fA_z =( &d_fv_z[first_i]);

    while(wtx < 100){

      rA_shared_v[wtx] =( rA_v[wtx]);

      rA_shared_x[wtx] =( rA_x[wtx]);

      rA_shared_y[wtx] =( rA_y[wtx]);

      rA_shared_z[wtx] =( rA_z[wtx]);

      wtx = wtx + 640;

    }

    wtx = tx;

    __syncthreads();

    for (k = 0; k < 1 + d_box_gpu[bx] . nn; k++) {

      if (k == 0) {

        pointer = bx;

      }

       else {

        pointer = d_box_gpu[bx] . nei[k - 1] . number;

      }

      first_j = d_box_gpu[pointer] . offset;

      rB_v =( &d_rv_v[first_j]);

      rB_x =( &d_rv_x[first_j]);

      rB_y =( &d_rv_y[first_j]);

      rB_z =( &d_rv_z[first_j]);

      qB =( &d_qv_gpu[first_j]);

      while(wtx < 100){

        rB_shared_v[wtx] =( rB_v[wtx]);

        rB_shared_x[wtx] =( rB_x[wtx]);

        rB_shared_y[wtx] =( rB_y[wtx]);

        rB_shared_z[wtx] =( rB_z[wtx]);

        qb_shared[wtx] =( qB[wtx]);

        wtx = wtx + 640;

      }

      wtx = tx;

      __syncthreads();

      while(wtx < 100){

        for (j = 0; j < 100; j++) {

          r2 = rA_shared_v[wtx] + rB_shared_v[j] - (rA_shared_x[wtx] * rB_shared_x[j] + rA_shared_y[wtx] * rB_shared_y[j] + rA_shared_z[wtx] * rB_shared_z[j]);

          u2 = a2 * r2;

          vij = exp(-u2);

          fs =( 2 * vij);

          s_x = rA_shared_x[wtx] - rB_shared_x[j];

          fxij =( fs * s_x);

          s_y = rA_shared_y[wtx] - rB_shared_y[j];

          fyij =( fs * s_y);

          s_z = rA_shared_z[wtx] - rB_shared_z[j];

          fzij =( fs * s_z);

          fA_v[wtx] =          fA_v[wtx]  +  qb_shared[j] * vij;

          fA_x[wtx] =          fA_x[wtx]  +  qb_shared[j] * fxij;

          fA_y[wtx] =          fA_y[wtx]  +  qb_shared[j] * fyij;

          fA_z[wtx] =          fA_z[wtx]  +  qb_shared[j] * fzij;

        }

        wtx = wtx + 640;

      }

      wtx = tx;

      __syncthreads();

    }

  }

}

int main(int argc,char *argv[])

{

  printf("thread block size of kernel = %d \n",640);

  int i;

  int j;

  int k;

  int l;

  int m;

  int n;

  float par_cpu;

  ::dim_str dim_cpu;

  ::box_str *box_cpu;

  int nh;

  dim_cpu . boxes1d_arg = 1;

  for (dim_cpu . cur_arg = 1; dim_cpu . cur_arg < argc; dim_cpu . cur_arg++) {

    if (strcmp(argv[dim_cpu . cur_arg],"-boxes1d") == 0) {

      if (argc >= dim_cpu . cur_arg + 1) {

        if (isInteger(argv[dim_cpu . cur_arg + 1]) == 1) {

          dim_cpu . boxes1d_arg = atoi(argv[dim_cpu . cur_arg + 1]);

          if (dim_cpu . boxes1d_arg < 0) {

            printf("ERROR: Wrong value to -boxes1d parameter, cannot be <=0\n");

            return 0;

          }

          dim_cpu . cur_arg = dim_cpu . cur_arg + 1;

        }

         else {

          printf("ERROR: Value to -boxes1d parameter in not a number\n");

          return 0;

        }

      }

       else {

        printf("ERROR: Missing value to -boxes1d parameter\n");

        return 0;

      }

    }

     else {

      printf("ERROR: Unknown parameter\n");

      return 0;

    }

  }

  printf("Configuration used: boxes1d = %d\n",dim_cpu . boxes1d_arg);

  par_cpu = 0.5;

  dim_cpu . number_boxes = (dim_cpu . boxes1d_arg * dim_cpu . boxes1d_arg * dim_cpu . boxes1d_arg);

  dim_cpu . space_elem = dim_cpu . number_boxes * 100;

  dim_cpu . box_mem = (dim_cpu . number_boxes * sizeof(::box_str ));

// allocate boxes

  box_cpu = ((::box_str *)(malloc(dim_cpu . box_mem)));

  nh = 0;

  for (i = 0; i < dim_cpu . boxes1d_arg; i++) {

// home boxes in y direction

    for (j = 0; j < dim_cpu . boxes1d_arg; j++) {

// home boxes in x direction

      for (k = 0; k < dim_cpu . boxes1d_arg; k++) {

// current home box

        box_cpu[nh] . x = k;

        box_cpu[nh] . y = j;

        box_cpu[nh] . z = i;

        box_cpu[nh] . number = nh;

        box_cpu[nh] . offset = (nh * 100);

// initialize number of neighbor boxes

        box_cpu[nh] . nn = 0;

// neighbor boxes in z direction

        for (l = - 1; l < 2; l++) {

// neighbor boxes in y direction

          for (m = - 1; m < 2; m++) {

// neighbor boxes in x direction

            for (n = - 1; n < 2; n++) {

              if ((i + l >= 0 && j + m >= 0 && k + n >= 0) == true && (i + l < dim_cpu . boxes1d_arg && j + m < dim_cpu . boxes1d_arg && k + n < dim_cpu . boxes1d_arg) == true && (l == 0 && m == 0 && n == 0) == false) {

                box_cpu[nh] . nei[box_cpu[nh] . nn] . x = k + n;

                box_cpu[nh] . nei[box_cpu[nh] . nn] . y = j + m;

                box_cpu[nh] . nei[box_cpu[nh] . nn] . z = i + l;

                box_cpu[nh] . nei[box_cpu[nh] . nn] . number = box_cpu[nh] . nei[box_cpu[nh] . nn] . z * dim_cpu . boxes1d_arg * dim_cpu . boxes1d_arg + box_cpu[nh] . nei[box_cpu[nh] . nn] . y * dim_cpu . boxes1d_arg + box_cpu[nh] . nei[box_cpu[nh] . nn] . x;

                box_cpu[nh] . nei[box_cpu[nh] . nn] . offset = (box_cpu[nh] . nei[box_cpu[nh] . nn] . number * 100);

                box_cpu[nh] . nn = box_cpu[nh] . nn + 1;

              }

// neighbor boxes in x direction

            }

// neighbor boxes in y direction

          }

// neighbor boxes in z direction

        }

        nh = nh + 1;

// home boxes in x direction

      }

// home boxes in y direction

    }

// home boxes in z direction

  }

  float *rv_cpu_v = new float [dim_cpu . space_elem];

  float *rv_cpu_x = new float [dim_cpu . space_elem];

  float *rv_cpu_y = new float [dim_cpu . space_elem];

  float *rv_cpu_z = new float [dim_cpu . space_elem];

  for (i = 0; i < dim_cpu . space_elem; i = i + 1) {

    rv_cpu_v[i] = (rand() % 10 + 1) / 10.0;

    rv_cpu_x[i] = (rand() % 10 + 1) / 10.0;

    rv_cpu_y[i] = (rand() % 10 + 1) / 10.0;

    rv_cpu_z[i] = (rand() % 10 + 1) / 10.0;

  }

  float *qv_cpu = new float [dim_cpu . space_elem];

  for (i = 0; i < dim_cpu . space_elem; i = i + 1) {

    qv_cpu[i] = (rand() % 10 + 1) / 10.0;

  }

  float *fv_cpu_v = new float [dim_cpu . space_elem];

  float *fv_cpu_x = new float [dim_cpu . space_elem];

  float *fv_cpu_y = new float [dim_cpu . space_elem];

  float *fv_cpu_z = new float [dim_cpu . space_elem];

  for (i = 0; i < dim_cpu . space_elem; i = i + 1) {

    fv_cpu_v[i] = ((float )0.0);

    fv_cpu_x[i] = ((float )0.0);

    fv_cpu_y[i] = ((float )0.0);

    fv_cpu_z[i] = ((float )0.0);

  }

  ::box_str *d_box_gpu;

  float *d_rv_v;

  float *d_rv_x;

  float *d_rv_y;

  float *d_rv_z;

  float *d_qv_gpu;

  float *d_fv_v;

  float *d_fv_x;

  float *d_fv_y;

  float *d_fv_z;

  hipDeviceSynchronize();

  ::dim3 threads;

  ::dim3 blocks;

  blocks . x = dim_cpu . number_boxes;

  blocks . y = 1;

  threads . x = 640;

  threads . y = 1;

  hipMemcpyToSymbol((dev_par),(&par_cpu),sizeof(float ));

  hipMalloc((void **)(&d_box_gpu),dim_cpu . box_mem);

  hipMalloc((void **)(&d_rv_v),dim_cpu . space_elem * sizeof(float ));

  hipMalloc((void **)(&d_rv_x),dim_cpu . space_elem * sizeof(float ));

  hipMalloc((void **)(&d_rv_y),dim_cpu . space_elem * sizeof(float ));

  hipMalloc((void **)(&d_rv_z),dim_cpu . space_elem * sizeof(float ));

  hipMalloc((void **)(&d_qv_gpu),dim_cpu . space_elem * sizeof(float ));

  hipMalloc((void **)(&d_fv_v),dim_cpu . space_elem * sizeof(float ));

  hipMalloc((void **)(&d_fv_x),dim_cpu . space_elem * sizeof(float ));

  hipMalloc((void **)(&d_fv_y),dim_cpu . space_elem * sizeof(float ));

  hipMalloc((void **)(&d_fv_z),dim_cpu . space_elem * sizeof(float ));

  struct timeval start_t;

  struct timeval end_t;

  struct timeval skt_t;

  struct timeval ske_t;

  gettimeofday(&start_t,0L);

  hipMemcpy(d_box_gpu,box_cpu,dim_cpu . box_mem,hipMemcpyHostToDevice);

  hipMemcpy(d_rv_v,rv_cpu_v,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  hipMemcpy(d_rv_x,rv_cpu_x,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  hipMemcpy(d_rv_y,rv_cpu_y,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  hipMemcpy(d_rv_z,rv_cpu_z,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  hipMemcpy(d_qv_gpu,qv_cpu,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  hipMemcpy(d_fv_v,fv_cpu_v,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  hipMemcpy(d_fv_x,fv_cpu_x,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  hipMemcpy(d_fv_y,fv_cpu_y,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  hipMemcpy(d_fv_z,fv_cpu_z,dim_cpu . space_elem * sizeof(float ),hipMemcpyHostToDevice);

  gettimeofday(&skt_t,0L);

  kernel_gpu_cuda<<<blocks,threads>>>(dim_cpu,d_box_gpu,d_rv_v,d_rv_x,d_rv_y,d_rv_z,d_qv_gpu,d_fv_v,d_fv_x,d_fv_y,d_fv_z);

  checkCUDAError("Start");

  hipDeviceSynchronize();

  gettimeofday(&ske_t,0L);

  hipMemcpy(fv_cpu_v,d_fv_v,dim_cpu . space_elem * sizeof(float ),hipMemcpyDeviceToHost);

  hipMemcpy(fv_cpu_x,d_fv_x,dim_cpu . space_elem * sizeof(float ),hipMemcpyDeviceToHost);

  hipMemcpy(fv_cpu_y,d_fv_y,dim_cpu . space_elem * sizeof(float ),hipMemcpyDeviceToHost);

  hipMemcpy(fv_cpu_z,d_fv_z,dim_cpu . space_elem * sizeof(float ),hipMemcpyDeviceToHost);

  gettimeofday(&end_t,0L);
  mpf_t val_x, val_y, val_in, err;
  mpf_init2(val_x, 128);
  mpf_init2(val_y, 128);
  mpf_init2(val_in, 128);
  mpf_init2(err, 128);
  FILE* infile = fopen("fv_ref.txt", "r");
  for(int i = 0; i < dim_cpu.space_elem; i++) {
    gmp_fscanf(infile, "%Fe\n", val_in);
    mpf_set_d(val_x, fv_cpu_v[i]);
    mpf_sub(val_y, val_x, val_in);
    mpf_abs(val_x, val_y);
    mpf_div(val_x, val_x, val_in);
    if (i==0)
      mpf_set(err, val_x);
    else
      mpf_add(err, err, val_x);
    gmp_fscanf(infile, "%Fe\n", val_in);
    mpf_set_d(val_x, fv_cpu_x[i]);
    mpf_sub(val_y, val_x, val_in);
    mpf_abs(val_x, val_y);
    mpf_div(val_x, val_x, val_in);
    mpf_add(err, err, val_x);
    gmp_fscanf(infile, "%Fe\n", val_in);
    mpf_set_d(val_x, fv_cpu_y[i]);
    mpf_sub(val_y, val_x, val_in);
    mpf_abs(val_x, val_y);
    mpf_div(val_x, val_x, val_in);
    mpf_add(err, err, val_x);
    gmp_fscanf(infile, "%Fe\n", val_in);
    mpf_set_d(val_x, fv_cpu_z[i]);
    mpf_sub(val_y, val_x, val_in);
    mpf_abs(val_x, val_y);
    mpf_div(val_x, val_x, val_in);
    mpf_add(err, err, val_x);
  }
  mpf_div_ui(err, err, 4*dim_cpu.space_elem);
  fclose(infile);
  gmp_printf("error: %10.5Fe\n", err);
  int blockSize;
  int minGridSize = dim_cpu.number_boxes;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel_gpu_cuda, 0, 0);
  printf("block: %d\n", blockSize);

  ((std::cout<<"time: ") << (end_t . tv_sec + end_t . tv_usec * 1e-6 - (start_t . tv_sec + start_t . tv_usec * 1e-6)))<<"\n";

  ((std::cout<<"kernel: ") << ((ske_t . tv_sec - skt_t . tv_sec) + (ske_t . tv_usec - skt_t . tv_usec) * 1e-6)) << endl;

  hipFree(d_rv_v);

  hipFree(d_rv_x);

  hipFree(d_rv_y);

  hipFree(d_rv_z);

  hipFree(d_qv_gpu);

  hipFree(d_fv_v);

  hipFree(d_fv_x);

  hipFree(d_fv_y);

  hipFree(d_fv_z);

  hipFree(d_box_gpu);

  free(rv_cpu_v);

  free(rv_cpu_x);

  free(rv_cpu_y);

  free(rv_cpu_z);

  free(qv_cpu);

  free(fv_cpu_v);

  free(fv_cpu_x);

  free(fv_cpu_y);

  free(fv_cpu_z);

  free(box_cpu);

  return 0;

}

