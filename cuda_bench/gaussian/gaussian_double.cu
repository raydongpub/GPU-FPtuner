#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <string.h>
#include <math.h>
#include <mpfr.h>
#include <iostream>

using namespace std;

#ifdef RD_WG_SIZE_0_0
        #define MAXBLOCKSIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define MAXBLOCKSIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define MAXBLOCKSIZE RD_WG_SIZE
#else
        #define MAXBLOCKSIZE 512
#endif

//2D defines. Go from specific to general                                                
#ifdef RD_WG_SIZE_1_0
        #define BLOCK_SIZE_XY RD_WG_SIZE_1_0
#elif defined(RD_WG_SIZE_1)
        #define BLOCK_SIZE_XY RD_WG_SIZE_1
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE_XY RD_WG_SIZE
#else
        #define BLOCK_SIZE_XY 4
#endif


FILE *fp;
unsigned int totalKernelTime = 0;

// create both matrix and right hand side, Ke Wang 2013/08/12 11:51:06
void
create_matrix(double *m, int size){
  int i,j;
  double lamda = -0.01;
  double coe[2*size-1];
  double coe_i =0.0;

  for (i=0; i < size; i++)
    {
      coe_i = 10*exp(lamda*i); 
      j=size-1+i;     
      coe[j]=coe_i;
      j=size-1-i;     
      coe[j]=coe_i;
    }


  for (i=0; i < size; i++) {
      for (j=0; j < size; j++) {
	m[i*size+j]=coe[size-1-i+j];
      }
  }


}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

/*-------------------------------------------------------
 ** Pay attention to the index.  Index i give the range
 ** which starts from 0 to range-1.  The real values of
 ** the index should be adjust and related with the value
 **-------------------------------------------------------
 */
__global__ void Fan1(double *m_cuda, double *a_cuda, int Size, int t)
{   

	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	m_cuda[Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t] = a_cuda[Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t] / a_cuda[Size*t+t];
	//*(m_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) = *(a_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) / *(a_cuda+Size*t+t);
}

/*-------------------------------------------------------
 **-------------------------------------------------------
 */ 

__global__ void Fan2(double *m_cuda, double *a_cuda, double *b_cuda,int Size, int j1, int t)
{
	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	if(threadIdx.y + blockIdx.y * blockDim.y >= Size-t) return;
	
	int xidx = blockIdx.x * blockDim.x + threadIdx.x;
	int yidx = blockIdx.y * blockDim.y + threadIdx.y;
	
	a_cuda[Size*(xidx+1+t)+(yidx+t)] -= m_cuda[Size*(xidx+1+t)+t] * a_cuda[Size*t+(yidx+t)];
	if(yidx == 0){
		b_cuda[xidx+1+t] -= m_cuda[Size*(xidx+1+t)+(yidx+t)] * b_cuda[t];
	}
}

/*------------------------------------------------------
 ** ForwardSub() -- Forward substitution of Gaussian
 ** elimination.
 **------------------------------------------------------
 */
/*------------------------------------------------------
 ** BackSub() -- Backward substitution
 **------------------------------------------------------
 */

int main(int argc, char *argv[])
{
  printf("WG size of kernel 1 = %d, WG size of kernel 2= %d X %d\n", MAXBLOCKSIZE, BLOCK_SIZE_XY, BLOCK_SIZE_XY);
    int verbose = 0;
    int i, j, t;
    char flag;
    if (argc < 2) {
        printf("Usage: gaussian -f filename / -s size [-q]\n\n");
        printf("-q (quiet) suppresses printing the matrix and result values.\n");
        printf("-f (filename) path of input file\n");
        printf("-s (size) size of matrix. Create matrix and rhs in this program \n");
        printf("The first line of the file contains the dimension of the matrix, n.");
        printf("The second line of the file is a newline.\n");
        printf("The next n lines contain n tab separated values for the matrix.");
        printf("The next line of the file is a newline.\n");
        printf("The next line of the file is a 1xn vector with tab separated values.\n");
        printf("The next line of the file is a newline. (optional)\n");
        printf("The final line of the file is the pre-computed solution. (optional)\n");
        printf("Example: matrix4.txt:\n");
        printf("4\n");
        printf("\n");
        printf("-0.6	-0.5	0.7	0.3\n");
        printf("-0.3	-0.9	0.3	0.7\n");
        printf("-0.4	-0.5	-0.3	-0.8\n");	
        printf("0.0	-0.1	0.2	0.9\n");
        printf("\n");
        printf("-0.85	-0.68	0.24	-0.53\n");	
        printf("\n");
        printf("0.7	0.0	-0.4	-0.5\n");
        exit(0);
    }
    

    int Size;
    double *a, *b, *finalVec;
    double *m;
    for(i=1;i<argc;i++) {
      if (argv[i][0]=='-') {// flag
        flag = argv[i][1];
          switch (flag) {
            case 's': // platform
              i++;
              Size = atoi(argv[i]);
	      printf("Create matrix internally in parse, size = %d \n", Size);

	      a = new double[Size * Size];
	      create_matrix(a, Size);

	      b = new double[Size];
	      for (j =0; j< Size; j++)
	    	b[j]=1.0;

	      m = new double[Size * Size];
              break;
	  }
      }
    }
	  finalVec = new double[Size];

    //InitProblemOnce(filename);
  	for (i=0; i<Size*Size; i++)
  			*(m+i) = (double)0.0;

    //begin timing
    struct timeval start_t;
    struct timeval end_t;
    struct timeval skt_t;
    struct timeval ske_t;
    struct timeval sht_t;
    struct timeval she_t;
    gettimeofday(&start_t,0L);
    
        double *m_cuda,*a_cuda,*b_cuda;

	// allocate memory on GPU
	hipMalloc((void **) &m_cuda, Size * Size * sizeof(double));
	hipMalloc((void **) &a_cuda, Size * Size * sizeof(double));
	hipMalloc((void **) &b_cuda, Size * sizeof(double));	
	// copy memory to GPU
	hipMemcpy(m_cuda, m, Size * Size * sizeof(double),hipMemcpyHostToDevice );
	hipMemcpy(a_cuda, a, Size * Size * sizeof(double),hipMemcpyHostToDevice );
	hipMemcpy(b_cuda, b, Size * sizeof(double),hipMemcpyHostToDevice );
	
	int block_size,grid_size;
	block_size = MAXBLOCKSIZE;
	grid_size = (Size/block_size) + (!(Size%block_size)? 0:1);
	//printf("1d grid size: %d\n",grid_size);
	dim3 dimBlock(block_size);
	dim3 dimGrid(grid_size);
	//dim3 dimGrid( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );
	int blockSize2d, gridSize2d;
	blockSize2d = BLOCK_SIZE_XY;
	gridSize2d = (Size/blockSize2d) + (!(Size%blockSize2d?0:1)); 
	dim3 dimBlockXY(blockSize2d,blockSize2d);
	dim3 dimGridXY(gridSize2d,gridSize2d);


  gettimeofday(&skt_t,0L);
	for (t=0; t<(Size-1); t++) {
		Fan1<<<dimGrid,dimBlock>>>(m_cuda,a_cuda,Size,t);
		hipDeviceSynchronize();
		Fan2<<<dimGridXY,dimBlockXY>>>(m_cuda,a_cuda,b_cuda,Size,Size-t,t);
		hipDeviceSynchronize();
		checkCUDAError("Fan2");
	}
  gettimeofday(&ske_t,0L);
	// copy memory back to CPU
	hipMemcpy(m, m_cuda, Size * Size * sizeof(double),hipMemcpyDeviceToHost );
	hipMemcpy(a, a_cuda, Size * Size * sizeof(double),hipMemcpyDeviceToHost );
	hipMemcpy(b, b_cuda, Size * sizeof(double),hipMemcpyDeviceToHost );

    //BackSub();
	// create a new vector to hold the final answer
	// solve "bottom up"
  gettimeofday(&sht_t,0L);
	for(i=0;i<Size;i++){
		finalVec[Size-i-1]=b[Size-i-1];
		for(j=0;j<i;j++)
		{
			finalVec[Size-i-1]-=a[Size*(Size-i-1)+(Size-j-1)] * finalVec[Size-j-1];
		}
		finalVec[Size-i-1]=finalVec[Size-i-1]/ a[Size*(Size-i-1)+(Size-i-1)];
	}
  gettimeofday(&she_t,0L);

  gettimeofday(&end_t,0L);


    ((std::cout<<"time: ") << ((end_t . tv_sec - start_t . tv_sec) + (end_t . tv_usec - start_t . tv_usec) * 1e-6)) << endl;
    ((std::cout<<"kernel: ") << ((ske_t . tv_sec - skt_t . tv_sec) + (ske_t . tv_usec - skt_t . tv_usec) * 1e-6 + (she_t . tv_sec - sht_t . tv_sec) + (she_t . tv_usec - sht_t . tv_usec) * 1e-6)) << endl;
    //if (verbose) {
    //    printf("The final solution is: \n");
    //    PrintAry(finalVec,Size);
    //}
    
	hipFree(m_cuda);
	hipFree(a_cuda);
	hipFree(b_cuda);
  free(m);
  free(a);
  free(b);
}
